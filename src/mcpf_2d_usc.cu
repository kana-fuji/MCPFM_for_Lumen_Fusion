#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////////
//
//  multicellular phase field model in 2D using cuda
//
/////////////////////////////////////////////////////////////////////////////////
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <iostream>
#include <fstream>
#include <sstream>

#include <iomanip> //setprecision 
#include <time.h>
#include "SFMT.h"
//#include <hiprand.h>
//#include <hiprand/hiprand_kernel.h>

#include <omp.h>

#include "config.cuh"
using namespace std;

const int WARP = 32; //number of threads in a wArp

const int IMAX = 2048;
const int JMAX = 2048;

const int CIMAX = 512;
const int CJMAX = 512;

const float DX = 0.01f;
const float DY = 0.01f;

const float T     =  900000.0f;
const float TOUT  =     100.0f;
const float TOUT2 =     100.0f;
const float DT    =       0.02f;
const float DTS   =       1.0f;

const int NMAX = 64;

PARAM::psys  h_sys;
PARAM::param h_para;
PARAM::cells h_cells[NMAX];
PARAM::dtv   h_dtv[NMAX];
PARAM::com   h_ijk[NMAX];
float h_xi;

__constant__ PARAM::psys  d_sys;
__constant__ PARAM::param d_para;
__constant__ PARAM::cells d_cells[NMAX];
__constant__ PARAM::com   d_ijk[NMAX];
__constant__ float d_xi;
__constant__ int   d_cnum;


//Mersenne twister---------------------------
sfmt_t sfmt;
void my_srand( uint32_t seed ) {
    sfmt_init_gen_rand( &sfmt, seed );
}
float MT_rand(){
  return (float)sfmt_genrand_res53(&sfmt);
}
//Mersenne twister---------------------------end


//Quick sort---------------------------
typedef float value_type; //type of key for sort

//return median of x, y, z
value_type med3(value_type x, value_type y, value_type z) {
  if (x < y) {
    if (y < z) return y; else if (z < x) return x; else return z;
  } else {
    if (z < y) return y; else if (x < z) return x; else return z;
  }
}

//quicksort
// a     : array
// left  : start position of array
// right : end position of array
void quicksort(value_type a[], int left, int right) {
  if (left < right) {
    int i = left, j = right;
    value_type tmp, pivot = med3(a[i], a[i + (j - i) / 2], a[j]); // overfrow on (i+j)/2
    while (1) { //devide a[] for clasters pivot over and less
      while (a[i] < pivot) i++; //surch a[i] >= pivot 
      while (pivot < a[j]) j--; //surch a[j] <= pivot
      if (i >= j) break;
      tmp = a[i]; a[i] = a[j]; a[j] = tmp; // change a[i] and a[j] 
      i++; j--;
    }
    quicksort(a, left, i - 1);  //sort left
    quicksort(a, j + 1, right); //sort right
  }
}
//Quick sort---------------------------end

////////////////////////////
//
//  Kernel function
//
////////////////////////////

__global__
void init_u_Kernel(float* u,float* r0){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % d_sys.cimax;
  int y = (index / d_sys.cimax) % d_sys.cjmax;
  int n = ((index / d_sys.cimax) / d_sys.cjmax) % d_cnum;


  float center_x=d_sys.cjmax*0.5f*d_sys.dx;
  float center_y=d_sys.cjmax*0.5f*d_sys.dy;
  float r;

  float xx,yy;

  xx=__fmul_rn(__fsub_rn(__fmul_rn(__fadd_rn(__int2float_rn(x),0.5f),d_sys.dx),center_x),
	       __fsub_rn(__fmul_rn(__fadd_rn(__int2float_rn(x),0.5f),d_sys.dx),center_x));
  yy=__fmul_rn(__fsub_rn(__fmul_rn(__fadd_rn(__int2float_rn(y),0.5f),d_sys.dx),center_y),
	       __fsub_rn(__fmul_rn(__fadd_rn(__int2float_rn(y),0.5f),d_sys.dx),center_y));

  r=__fsqrt_rn(__fadd_rn(xx,yy));
  u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]=__fmul_rn(__fsub_rn(1.0f,
								   tanhf(__fdiv_rn(__fsub_rn(r,r0[n]),
										   __fsqrt_rn(2.0f*d_para.D_u)))),
							 0.5f);
}



__device__
float h(const float pf){
  return __fmul_rn(__fmul_rn(pf,pf),(__fsub_rn(3.0f,2.0f*pf)));
}



__device__
float laplacian4(const float v1,
		 const float vx0,const float vx2,
		 const float vy0,const float vy2
		 ){
  return __fdiv_rn(__fadd_rn(__fsub_rn(__fadd_rn(vx2,vx0),__fadd_rn(v1,v1)),
			     __fsub_rn(__fadd_rn(vy2,vy0),__fadd_rn(v1,v1))),
		   __fmul_rn(d_sys.dx,d_sys.dx));
}


__device__
float laplacian8(const float v1,
		 const float vx0,const float vx2,const float vy0,const float vy2,
		 const float vx0y0,const float vx2y0,const float vx0y2,const float vx2y2
		 ){

  // return __fmul_rn(__fadd_rn(
  //  			     __fadd_rn(
  //  				       __fdiv_rn(__fsub_rn(__fadd_rn(vx2,vx0),__fadd_rn(v1,v1)),
  //  						 __fmul_rn(d_sys.dx,d_sys.dx)),
  //  				       __fdiv_rn(__fsub_rn(__fadd_rn(vy2,vy0),__fadd_rn(v1,v1)),
  //  						 __fmul_rn(d_sys.dx,d_sys.dx))
  //  				       ),
  //  			     __fdiv_rn(__fsub_rn(__fmul_rn(__fadd_rn(__fadd_rn(vx2y0,vx0y0),
  //  								     __fadd_rn(vx0y2,vx2y2)),
  //  							   0.5f),
  //  						 __fmul_rn(2.0f,v1)),
  //  				       __fmul_rn(d_sys.dx,d_sys.dx))
  //  			     ),
  //  		   0.5f);
  return
    ((vx0-2.0f*v1+vx2)/(d_sys.dx*d_sys.dx)
     +(vy0-2.0f*v1+vy2)/(d_sys.dy*d_sys.dy)
     +(0.5f*(vx0y0+vx2y0+vx0y2+vx2y2)-2.0f*v1)/(d_sys.dy*d_sys.dy)
     )*0.5f;

}






__global__
void time_evolution_Kernel(float** pf_temp,float** pf,float *phi,int row){

  int n = blockIdx.y;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cjmax;
  int x = (index / d_sys.cjmax) % d_sys.cimax;


  if(x>0 && x<d_sys.cimax-1 && y>0 && y<d_sys.cimax-1){
    pf_temp[n][x+d_sys.cimax*y]
      =pf[n][x+d_sys.cimax*y]
      +(
	//d_para.D_u*laplacian8(x,y,row,pf[n])
	d_para.D_u*(
		    (pf[n][x+row*(y-1)]-2.00f*pf[n][x+row*y]+pf[n][x+row*(y+1)])
		    /(d_sys.dx*d_sys.dx)
		    +(pf[n][(x-1)+row*y]-2.00f*pf[n][x+row*y]+pf[n][(x+1)+row*y])
		    /(d_sys.dy*d_sys.dy)
		    +(0.50f*
		      (pf[n][(x-1)+row*(y-1)]+pf[n][(x-1)+row*(y+1)]
		       +pf[n][(x+1)+row*(y-1)]+pf[n][(x+1)+row*(y+1)])-2.0f*pf[n][x+row*y])
		    /(d_sys.dx*d_sys.dy)
		    )*0.50f
	+pf[n][x+d_sys.cimax*y]*(1-pf[n][x+d_sys.cimax*y])
	*(
	  pf[n][x+d_sys.cimax*y]-0.50f
	  //+d_para.alpha*(d_para.V-d_cells[n].v)
	  +d_para.alpha*(d_cells[n].targetv-d_cells[n].v)
	  -d_para.beta*(phi[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]-h(pf[n][x+d_sys.cimax*y]))
	  +d_para.eta*(
		       (
			(phi[(x+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]
			 -h(pf[n][x+d_sys.cimax*(y-1)]))
			-2.00f*(phi[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
				-h(pf[n][x+d_sys.cimax*y]))
			+(phi[(x+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]
			 -h(pf[n][x+d_sys.cimax*(y+1)]))
			)
		       /(d_sys.dx*d_sys.dx)
		       +(
			 (phi[((x-1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
			  -h(pf[n][(x-1)+d_sys.cimax*y]))
			-2.00f*(phi[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
				-h(pf[n][x+d_sys.cimax*y]))
			 +(phi[((x+1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
			   -h(pf[n][(x+1)+d_sys.cimax*y]))
			 )
		       /(d_sys.dy*d_sys.dy)
		       +(0.50f*
			 (
			  (phi[((x-1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]
			   -h(pf[n][(x-1)+d_sys.cimax*(y-1)]))
			  +(phi[((x-1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]
			   -h(pf[n][(x-1)+d_sys.cimax*(y+1)]))
			  +(phi[((x+1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]
			   -h(pf[n][(x+1)+d_sys.cimax*(y-1)]))
			  +(phi[((x+1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]
			   -h(pf[n][(x+1)+d_sys.cimax*(y+1)]))
			  )
			 -2.0f*(phi[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
				-h(pf[n][x+d_sys.cimax*y]))
			 )
		       /(d_sys.dx*d_sys.dy)
		       )*0.50f
	  +d_para.gamma*(
			 (pf[n][x+row*(y-1)]-2.00f*pf[n][x+row*y]+pf[n][x+row*(y+1)])
			 /(d_sys.dx*d_sys.dx)
			 +(pf[n][(x-1)+row*y]-2.00f*pf[n][x+row*y]+pf[n][(x+1)+row*y])
			 /(d_sys.dy*d_sys.dy)
			 +(0.50f*
			   (pf[n][(x-1)+row*(y-1)]+pf[n][(x-1)+row*(y+1)]
			    +pf[n][(x+1)+row*(y-1)]+pf[n][(x+1)+row*(y+1)])-2.0f*pf[n][x+row*y])
			 /(d_sys.dx*d_sys.dy)
			 )*0.50f
	  )
	)*d_sys.dt/d_para.tau_u
      ;
  }
}


__global__
void time_evolution_with_reshaping_Kernel(float** pf_temp,float** pf,float *phi,float *c,int row){

  int n = blockIdx.y;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cjmax;
  int x = (index / d_sys.cjmax) % d_sys.cimax;


  if(x>0 && x<d_sys.cimax-1 && y>0 && y<d_sys.cimax-1){
    pf_temp[n][x+d_sys.cimax*y]
      =pf[n][x+d_sys.cimax*y]
      +(
	pf[n][x+d_sys.cimax*y]*(1-pf[n][x+d_sys.cimax*y])
	*(
	  //d_para.alpha*(d_para.V-d_cells[n].v)
	  +d_para.alpha*(d_cells[n].targetv-d_cells[n].v)
	  -d_para.beta*(phi[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]-h(pf[n][x+d_sys.cimax*y]))
	  +d_para.eta*(
		       (
			(phi[(x+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]
			 -h(pf[n][x+d_sys.cimax*(y-1)]))
			-2.00f*(phi[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
				-h(pf[n][x+d_sys.cimax*y]))
			+(phi[(x+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]
			 -h(pf[n][x+d_sys.cimax*(y+1)]))
			)
		       /(d_sys.dx*d_sys.dx)
		       +(
			 (phi[((x-1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
			  -h(pf[n][(x-1)+d_sys.cimax*y]))
			-2.00f*(phi[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
				-h(pf[n][x+d_sys.cimax*y]))
			 +(phi[((x+1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
			   -h(pf[n][(x+1)+d_sys.cimax*y]))
			 )
		       /(d_sys.dy*d_sys.dy)
		       +(0.50f*
			 (
			  (phi[((x-1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]
			   -h(pf[n][(x-1)+d_sys.cimax*(y-1)]))
			  +(phi[((x-1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]
			   -h(pf[n][(x-1)+d_sys.cimax*(y+1)]))
			  +(phi[((x+1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]
			   -h(pf[n][(x+1)+d_sys.cimax*(y-1)]))
			  +(phi[((x+1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]
			   -h(pf[n][(x+1)+d_sys.cimax*(y+1)]))
			  )
			 -2.0f*(phi[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
				-h(pf[n][x+d_sys.cimax*y]))
			 )
		       /(d_sys.dx*d_sys.dy)
		       )*0.50f
	  )
	)*d_sys.dt/d_para.tau_u
      +2.0f*d_para.gamma_curv*sqrtf(d_para.D_u)
      *(
	(pf[n][x+row*(y-1)]-2.00f*pf[n][x+row*y]+pf[n][x+row*(y+1)])
	/(d_sys.dx*d_sys.dx)
	+(pf[n][(x-1)+row*y]-2.00f*pf[n][x+row*y]+pf[n][(x+1)+row*y])
	/(d_sys.dy*d_sys.dy)
	+(0.50f*
	  (pf[n][(x-1)+row*(y-1)]+pf[n][(x-1)+row*(y+1)]
	   +pf[n][(x+1)+row*(y-1)]+pf[n][(x+1)+row*(y+1)])
	  -2.0f*pf[n][x+row*y]
	  )/(d_sys.dx*d_sys.dy)
	)*0.50f
      *d_sys.dt/d_para.tau_u
      ;

    //interaction with medium
    pf_temp[n][x+d_sys.cimax*y]
      +=pf[n][x+d_sys.cimax*y]*(1-pf[n][x+d_sys.cimax*y])
      *(
	-d_para.beta_cu*c[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
	+d_para.eta_cu*(
			(c[(x+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]
			 -2.00f*c[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
			 +c[(x+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]
			 )/(d_sys.dx*d_sys.dx)
		       +(c[((x-1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
			 -2.00f*c[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
			 +c[((x+1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
			 )/(d_sys.dy*d_sys.dy)
		       +(0.50f*
			 (
			  c[((x-1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]
			  +c[((x-1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]
			  +c[((x+1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]
			  +c[((x+1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]
			  )
			 -2.0f*c[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
			 )/(d_sys.dx*d_sys.dy)
			)*0.50f
	)*d_sys.dt/d_para.tau_u
      ;
  }
}


__global__
void time_evolution_u_with_reshaping_Kernel(float* u_temp,float* u,float *phi,float *s,float *c){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % d_sys.cimax;
  int y = (index / d_sys.cimax) % d_sys.cjmax;
  int n = ((index / d_sys.cimax) / d_sys.cjmax) % d_cnum;

  float dttau=__fdiv_rn(d_sys.dt,d_para.tau_u);

  if(x>0 && x<d_sys.cimax-1 && y>0 && y<d_sys.cimax-1){
    //cells
    u_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
      =u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
      +(
	 //diffusion
	 // d_para.D_u*laplacian8(u[n][x+row*y],
	 // 			  u[n][(x-1)+row*y],u[n][(x+1)+row*y],
	 // 			  u[n][x+row*(y-1)],u[n][x+row*(y+1)],
	 // 			  u[n][(x-1)+row*(y-1)],u[n][(x+1)+row*(y-1)],
	 // 			  u[n][(x-1)+row*(y+1)],u[n][(x+1)+row*(y+1)])

	 u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
	 *(1.0f-u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax])
	 *(
	   //double well potential
	   //(u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]-0.50f)

	   //volume conservation
	   //+d_para.alpha*(d_para.V-d_cells[n].v)
	   d_para.alpha*(d_cells[n].targetv-d_cells[n].v)


	   //excluded volume
	   -1.0f*d_para.beta
	   *(phi[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
	     -h(u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax])
	     )
			
	   //cell-cell adhesion
	   +d_para.eta*
	   laplacian8(phi[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
		      -h(u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]),
		      phi[((x-1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
		      -h(u[(x-1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]),
		      phi[((x+1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
		      -h(u[(x+1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]),
		      phi[(x+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]
		      -h(u[x+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cjmax]),
		      phi[(x+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]
		      -h(u[x+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cjmax]),
		      phi[((x-1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]
		      -h(u[(x-1)+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cjmax]),
		      phi[((x+1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]
		      -h(u[(x+1)+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cjmax]),
		      phi[((x-1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]
		      -h(u[(x-1)+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cjmax]),
		      phi[((x+1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]
		      -h(u[(x+1)+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cjmax]))

	   //curveture effect
	   +12.0f*d_para.gamma_curv*__fsqrt_rn(d_para.D_u)*
	   laplacian8(h(u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]),
	    	      h(u[(x-1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]),
	    	      h(u[(x+1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]),
	    	      h(u[x+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cjmax]),
	    	      h(u[x+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cjmax]),
	    	      h(u[(x-1)+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cjmax]),
	    	      h(u[(x+1)+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cjmax]),
	    	      h(u[(x-1)+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cjmax]),
	    	      h(u[(x+1)+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cjmax]))

	   )
	 )*dttau;

    //interaction with lumen
    u_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
      +=u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
      *(1.0f-u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax])
      *(
    	-d_para.beta_s*h(s[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)])
    	+d_para.eta_s*laplacian8(h(s[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]),
    				 h(s[((x-1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]),
    				 h(s[((x+1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]),
    				 h(s[(x+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]),
    				 h(s[(x+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]),
    				 h(s[((x-1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]),
    				 h(s[((x+1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]),
    				 h(s[((x-1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]),
    				 h(s[((x+1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]))
    	)*dttau;

    //interaction with medium
    u_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
      +=u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
      *(1.0f-u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax])
      *(
    	-d_para.beta_cu*h(c[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)])
    	+d_para.eta_cu*laplacian8(h(c[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]),
				  h(c[((x-1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]),
				  h(c[((x+1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]),
				  h(c[(x+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]),
				  h(c[(x+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]),
				  h(c[((x-1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]),
				  h(c[((x+1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]),
				  h(c[((x-1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]),
				  h(c[((x+1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]))
    	)*dttau;

  }
}



__global__
void time_evolution_up_with_reshaping_Kernel_test(float *u_temp,float *u,
						  float *p_temp,float *p,
						  float *phi,float*pall,float *s,float *c,
						  float *u_adhe){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % d_sys.cimax;
  int y = (index / d_sys.cimax) % d_sys.cjmax;
  int n = ((index / d_sys.cimax) / d_sys.cjmax) % d_cnum;

  float dttau=__fdiv_rn(d_sys.dt,d_para.tau_u);

  if(x>0 && x<d_sys.cimax-1 && y>0 && y<d_sys.cimax-1){
    //cells
    u_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
      =u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
      +(
	 //diffusion
	 // d_para.D_u*laplacian8(u[n][x+row*y],
	 // 			  u[n][(x-1)+row*y],u[n][(x+1)+row*y],
	 // 			  u[n][x+row*(y-1)],u[n][x+row*(y+1)],
	 // 			  u[n][(x-1)+row*(y-1)],u[n][(x+1)+row*(y-1)],
	 // 			  u[n][(x-1)+row*(y+1)],u[n][(x+1)+row*(y+1)])

	 u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
	 *(1.0f-u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax])
	 *(
	   //double well potential
	   //(u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]-0.50f)

	   //volume conservation
	   //+d_para.alpha*(d_para.V-d_cells[n].v)
	   d_para.alpha*(d_cells[n].targetv-d_cells[n].v)


	   //excluded volume
	   -1.0f*d_para.beta
	   *(phi[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
	     -h(u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax])
	     )
			

	   //cell-cell adhesion with anti-adhesion term 1
	   +d_para.eta/2.0f*(1+tanhf((d_para.p_th
				     -pall[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)])
				    /d_para.l_anti))
	   *laplacian8(phi[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
		       -h(u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]),
		       phi[((x-1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
		       -h(u[(x-1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]),
		       phi[((x+1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
		       -h(u[(x+1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]),
		       phi[(x+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]
		       -h(u[x+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cjmax]),
		       phi[(x+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]
		       -h(u[x+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cjmax]),
		       phi[((x-1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]
		       -h(u[(x-1)+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cjmax]),
		       phi[((x+1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]
		       -h(u[(x+1)+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cjmax]),
		       phi[((x-1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]
		       -h(u[(x-1)+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cjmax]),
		       phi[((x+1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]
		       -h(u[(x+1)+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cjmax]))


	   //cell-cell adhesion with anti-adhesion term 2
	    +(
	      (pall[((x+1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]-
	       pall[((x-1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)])
	      /(2.0f*d_sys.dx)
	      *((phi[((x+1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
	    	-h(u[(x+1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]))-
	        (phi[((x-1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]
	    	-h(u[(x-1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax])))
	      /(2.0f*d_sys.dx)
	      +
	      (pall[(x+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]-
	       pall[(x+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)])
	      /(2.0f*d_sys.dy)
	      *((phi[(x+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]
	    	-h(u[x+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cjmax]))-
	        (phi[(x+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]
	    	-h(u[x+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cjmax])))
	      /(2.0f*d_sys.dy)
	      )
	    *d_para.eta/(4.0f*d_para.l_anti
	    		*cosh((d_para.p_th-pall[(x+d_cells[n].cimin)
	    					+d_sys.imax*(y+d_cells[n].cjmin)])/d_para.l_anti)
	    		*cosh((d_para.p_th-pall[(x+d_cells[n].cimin)
	    					+d_sys.imax*(y+d_cells[n].cjmin)])/d_para.l_anti))


	   //curveture effect
	   +12.0f*d_para.gamma_curv*__fsqrt_rn(d_para.D_u)*
	   laplacian8(u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax],
	    	      u[(x-1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax],
	    	      u[(x+1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax],
	    	      u[x+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cjmax],
	    	      u[x+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cjmax],
	    	      u[(x-1)+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cjmax],
	    	      u[(x+1)+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cjmax],
	    	      u[(x-1)+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cjmax],
	    	      u[(x+1)+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cjmax])

	   )
	 )*dttau;

    //interaction with lumen
    u_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
      +=u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
      *(1.0f-u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax])
      *(
    	-d_para.beta_s*h(s[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)])
    	+d_para.eta_s*laplacian8(h(s[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]),
    				 h(s[((x-1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]),
    				 h(s[((x+1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]),
    				 h(s[(x+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]),
    				 h(s[(x+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]),
    				 h(s[((x-1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]),
    				 h(s[((x+1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]),
    				 h(s[((x-1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]),
    				 h(s[((x+1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]))
    	)*dttau;

    //interaction with medium
    u_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
      +=u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
      *(1.0f-u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax])
      *(
    	-d_para.beta_cu*h(c[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)])
    	+d_para.eta_cu*laplacian8(h(c[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]),
				  h(c[((x-1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]),
				  h(c[((x+1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]),
				  h(c[(x+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]),
				  h(c[(x+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]),
				  h(c[((x-1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]),
				  h(c[((x+1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]),
				  h(c[((x-1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]),
				  h(c[((x+1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]))
    	)*dttau;



    //anti-adhesive molecules
    p_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
      =p[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
      +(
	//diffusion
	d_para.D_p*laplacian8(p[x+y*d_sys.cimax+n*d_sys.cimax*d_sys.cjmax],
			      p[(x-1)+y*d_sys.cimax+n*d_sys.cimax*d_sys.cjmax],
			      p[(x+1)+y*d_sys.cimax+n*d_sys.cimax*d_sys.cjmax],
			      p[x+(y-1)*d_sys.cimax+n*d_sys.cimax*d_sys.cjmax],
			      p[x+(y+1)*d_sys.cimax+n*d_sys.cimax*d_sys.cjmax],
			      p[(x-1)+(y-1)*d_sys.cimax+n*d_sys.cimax*d_sys.cjmax],
			      p[(x+1)+(y-1)*d_sys.cimax+n*d_sys.cimax*d_sys.cjmax],
			      p[(x-1)+(y+1)*d_sys.cimax+n*d_sys.cimax*d_sys.cjmax],
			      p[(x+1)+(y+1)*d_sys.cimax+n*d_sys.cimax*d_sys.cjmax])

	//interface
	+p[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
	*(1.0f-p[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax])
	*(
	  //double well potential
	  d_para.k_p*(p[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]-0.50f)

	  // //curveture effect
	  // +d_para.gamma_p
	  // *laplacian8(p[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax],
	  //   	      p[(x-1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax],
	  //   	      p[(x+1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax],
	  //   	      p[x+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cjmax],
	  //   	      p[x+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cjmax],
	  //   	      p[(x-1)+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cjmax],
	  //   	      p[(x+1)+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cjmax],
	  //   	      p[(x-1)+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cjmax],
	  //   	      p[(x+1)+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cjmax])
	  )

	//cell-cell adhesion with anti-adhesion
	+u_adhe[(x+d_cells[n].cimin)]//d_para.eta/6.0f*\nabla h(u_m).\nabla h(u_n)
	/(4.0f*d_para.l_anti
	  *cosh((d_para.p_th-pall[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)])
		/d_para.l_anti)
	  *cosh((d_para.p_th-pall[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)])
		/d_para.l_anti))

	//volume conservation
	+d_para.alpha_p*(d_para.Vp-d_cells[n].vp)

	//localization on cell membrane
	+d_para.C_p*u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
	*(1.0f-u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax])

        //apical-lumen adhesion
	+d_para.eta_ps/6.0f
	*laplacian8(h(s[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]),
		    h(s[((x-1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]),
		    h(s[((x+1)+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]),
		    h(s[(x+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]),
		    h(s[(x+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]),
		    h(s[((x-1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]),
		    h(s[((x+1)+d_cells[n].cimin)+d_sys.imax*((y-1)+d_cells[n].cjmin)]),
		    h(s[((x-1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]),
		    h(s[((x+1)+d_cells[n].cimin)+d_sys.imax*((y+1)+d_cells[n].cjmin)]))

	)*d_sys.dt/d_para.tau_p
      ;
  }
}





__global__
void time_evolution_c_with_reshaping_Kernel(float* pf_temp,float* pf,float *phi,int row){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.jmax;
  int x = (index / d_sys.jmax) % d_sys.imax;


  if(x>0 && x<d_sys.imax-1 && y>0 && y<d_sys.imax-1){
    pf_temp[x+d_sys.imax*y]
      =pf[x+d_sys.imax*y]
      +(
	pf[x+d_sys.imax*y]*(1-pf[x+d_sys.imax*y])
	*(
	  d_para.xi_c
	  -d_para.beta_cu*phi[x+d_sys.imax*y]
	  +d_para.eta_cu*(
			  (phi[x+d_sys.imax*(y-1)]-2.00f*phi[x+d_sys.imax*y]+phi[x+d_sys.imax*(y+1)]
			   )/(d_sys.dx*d_sys.dx)
			  +(phi[(x-1)+d_sys.imax*y]-2.00f*phi[x+d_sys.imax*y]+phi[(x+1)+d_sys.imax*y]
			    )/(d_sys.dy*d_sys.dy)
			  +(0.50f*
			    (phi[(x-1)+d_sys.imax*(y-1)]+phi[(x-1)+d_sys.imax*(y+1)]
			     +phi[(x+1)+d_sys.imax*(y-1)]+phi[(x+1)+d_sys.imax*(y+1)])
			    -2.0f*phi[x+d_sys.imax*y]
			    )/(d_sys.dx*d_sys.dy)
			  )*0.50f
	  )
	)*d_sys.dt/d_para.tau_c
      ;
  }
}



__global__
void time_evolution_sc_with_reshaping_Kernel(float* c_temp,float* c,float* s_temp,float* s,
					     float *phi,float* uall,float* pall){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.jmax;
  int x = (index / d_sys.jmax) % d_sys.imax;


  if(x>0 && x<d_sys.imax-1 && y>0 && y<d_sys.jmax-1){
    //medium
    c_temp[x+d_sys.imax*y]
      =c[x+d_sys.imax*y]
      +(
	c[x+d_sys.imax*y]*(1.0f-c[x+d_sys.imax*y])
	*(
	  //pressuer
	  d_para.xi_c

	  //excluded volume with cells
	  -d_para.beta_cu*phi[x+d_sys.imax*y]

	  //excluded volume with lumen
	  -d_para.beta_cs*h(s[x+d_sys.imax*y])

	  //cell-medium adhesion
	  +d_para.eta_cu*laplacian8(phi[x+d_sys.imax*y],
				    phi[(x-1)+d_sys.imax*y],phi[(x+1)+d_sys.imax*y],
				    phi[x+d_sys.imax*(y-1)],phi[x+d_sys.imax*(y+1)],
				    phi[(x-1)+d_sys.imax*(y-1)],phi[(x+1)+d_sys.imax*(y-1)],
				    phi[(x-1)+d_sys.imax*(y+1)],phi[(x+1)+d_sys.imax*(y+1)])


	  )
	)*d_sys.dt/d_para.tau_c
      ;

    //lumen
    s_temp[x+d_sys.imax*y]
      =s[x+d_sys.imax*y]
      +(
	s[x+d_sys.imax*y]*(1.0f-s[x+d_sys.imax*y])
	*(
	  //pressuer
	  d_para.xi
	  //d_xi

	  //excluded volume with cells
	  -d_para.beta_s*phi[x+d_sys.imax*y]

	  //excluded volume with medium
	  -d_para.beta_cs*h(c[x+d_sys.imax*y])

	  //cell-lumen adhesion
	  +d_para.eta_s*laplacian8(phi[x+d_sys.imax*y],
				   phi[(x-1)+d_sys.imax*y],phi[(x+1)+d_sys.imax*y],
				   phi[x+d_sys.imax*(y-1)],phi[x+d_sys.imax*(y+1)],
				   phi[(x-1)+d_sys.imax*(y-1)],phi[(x+1)+d_sys.imax*(y-1)],
				   phi[(x-1)+d_sys.imax*(y+1)],phi[(x+1)+d_sys.imax*(y+1)])

	  )
	)*d_sys.dt/d_para.tau_s
      ;

    //local volume conservation
    if(pall[x+d_sys.imax*y]>d_para.p_st &&
       s[x+d_sys.imax*y]+uall[x+d_sys.imax*y]+c[x+d_sys.imax*y]<d_para.v_t){
      s_temp[x+d_sys.imax*y]
	+=d_para.alpha_s*(1.0f-(s[x+d_sys.imax*y]+uall[x+d_sys.imax*y]+c[x+d_sys.imax*y]))
				*d_sys.dt/d_para.tau_s;
    }
  }
}


__global__
void time_evolution_sc_with_reshaping_2_Kernel(float* c_temp,float* c,float* s_temp,float* s,
					       float *phi,float* uall,
					       int imin,int jmin,int imax,int jmax,
					       float vc){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % imax;
  int y = (index / imax) % jmax;


  if(x+imin>0 && x+imin<d_sys.imax-1 && y+jmin>0 && y+jmin<d_sys.jmax-1){
    //medium
    c_temp[(x+imin)+d_sys.imax*(y+jmin)]
      =c[(x+imin)+d_sys.imax*(y+jmin)]
      +(
	c[(x+imin)+d_sys.imax*(y+jmin)]*(1.0f-c[(x+imin)+d_sys.imax*(y+jmin)])
	*(
	  //pressuer
	  d_para.xi_c

	  //excluded volume with cells
	  -d_para.beta_cu*phi[(x+imin)+d_sys.imax*(y+jmin)]

	  //excluded volume with lumen
	  -d_para.beta_cs*h(s[(x+imin)+d_sys.imax*(y+jmin)])

	  //cell-medium adhesion
	  +d_para.eta_cu*laplacian8(phi[(x+imin)+d_sys.imax*(y+jmin)],
				    phi[(x+imin-1)+d_sys.imax*(y+jmin)],phi[(x+imin+1)+d_sys.imax*(y+jmin)],
				    phi[(x+imin)+d_sys.imax*(y+jmin-1)],phi[(x+imin)+d_sys.imax*(y+jmin+1)],
				    phi[(x+imin-1)+d_sys.imax*(y+jmin-1)],phi[(x+imin+1)+d_sys.imax*(y+jmin-1)],
				    phi[(x+imin-1)+d_sys.imax*(y+jmin+1)],phi[(x+imin+1)+d_sys.imax*(y+jmin+1)])

          //20220520 Bulk modulus
          +d_para.alpha_c*(d_sys.imax*d_sys.jmax*d_sys.dx*d_sys.dy-vc)

	  )
	)*d_sys.dt/d_para.tau_c
      ;

    //lumen
    s_temp[(x+imin)+d_sys.imax*(y+jmin)]
      =s[(x+imin)+d_sys.imax*(y+jmin)]
      +(
	s[(x+imin)+d_sys.imax*(y+jmin)]*(1.0f-s[(x+imin)+d_sys.imax*(y+jmin)])
	*(
	  //pressuer
	  d_para.xi
	  //d_xi

	  //excluded volume with cells
	  -d_para.beta_s*phi[(x+imin)+d_sys.imax*(y+jmin)]

	  //excluded volume with medium
	  -d_para.beta_cs*h(c[(x+imin)+d_sys.imax*(y+jmin)])

	  //cell-lumen adhesion
	  +d_para.eta_s*laplacian8(phi[(x+imin)+d_sys.imax*(y+jmin)],
				   phi[(x+imin-1)+d_sys.imax*(y+jmin)],
				   phi[(x+imin+1)+d_sys.imax*(y+jmin)],
				   phi[(x+imin)+d_sys.imax*(y+jmin-1)],
				   phi[(x+imin)+d_sys.imax*(y+jmin+1)],
				   phi[(x+imin-1)+d_sys.imax*(y+jmin-1)],
				   phi[((x+imin)+1)+d_sys.imax*(y+jmin-1)],
				   phi[(x+imin-1)+d_sys.imax*(y+jmin+1)],
				   phi[((x+imin)+1)+d_sys.imax*(y+jmin+1)])

	  )
	)*d_sys.dt/d_para.tau_s
      ;

  }
}



__global__
void u_n_Kernel(float* u_n, float* u, int n){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;

  u_n[x+d_sys.cimax*y]=u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax];
}

__global__
void up_n_Kernel(float* u_n, float* u,float* p_n, float* p,int n){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;

  u_n[x+d_sys.cimax*y]=u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax];
  p_n[x+d_sys.cimax*y]=p[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax];
}


__global__
void update_Kernel(float* pf, float* pf_temp){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % d_sys.cimax;
  int y = (index / d_sys.cimax) % d_sys.cjmax;
  int n = ((index / d_sys.cimax) / d_sys.cjmax) % d_cnum;

  pf[x+y*d_sys.cimax+n*d_sys.cimax*d_sys.cjmax]=pf_temp[x+y*d_sys.cimax+n*d_sys.cimax*d_sys.cjmax];
}

__global__
void update_up_Kernel(float* u, float* u_temp,float* p, float* p_temp){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % d_sys.cimax;
  int y = (index / d_sys.cimax) % d_sys.cjmax;
  int n = ((index / d_sys.cimax) / d_sys.cjmax) % d_cnum;

  u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]=u_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax];
  p[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]=p_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax];
}


__global__
void update_all_Kernel(float* pf, float* pf_temp){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.imax;
  int x = (index / d_sys.imax) % d_sys.imax;

  pf[x+d_sys.imax*y]=pf_temp[x+d_sys.imax*y];
}

__global__
void update_all_2_Kernel(float* pf, float* pf_temp,int imin,int jmin,int imax,int jmax){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % imax;
  int y = (index / imax) % jmax;

  pf[(x+imin)+d_sys.imax*(y+jmin)]=pf_temp[(x+imin)+d_sys.imax*(y+jmin)];
}


__global__
void update_sc_Kernel(float* s, float* s_temp,float* c, float* c_temp){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.imax;
  int x = (index / d_sys.imax) % d_sys.imax;

  s[x+d_sys.imax*y]=s_temp[x+d_sys.imax*y];
  c[x+d_sys.imax*y]=c_temp[x+d_sys.imax*y];
}

__global__
void update_sc_2_Kernel(float* s, float* s_temp,float* c, float* c_temp,int imin,int jmin,int imax,int jmax){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % imax;
  int y = (index / imax) % jmax;

  s[(x+imin)+d_sys.imax*(y+jmin)]=s_temp[(x+imin)+d_sys.imax*(y+jmin)];
  c[(x+imin)+d_sys.imax*(y+jmin)]=c_temp[(x+imin)+d_sys.imax*(y+jmin)];
}


__global__
void CoM_Kernel(float** pf, float** pf_temp){
  int n = blockIdx.y;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;

  int ii,jj;
  ii=x+d_ijk[n].di; jj=y+d_ijk[n].dj;
  if(ii>0 && ii<d_sys.cimax-1 && jj>0 && jj<d_sys.cjmax-1)
    pf_temp[n][x+d_sys.cimax*y]=pf[n][ii+d_sys.cimax*jj];
  else
    pf_temp[n][x+d_sys.cimax*y]=0.0f;
}

__global__
void CoM_up_Kernel(float* u, float* u_temp,float* p, float* p_temp){
  int n = blockIdx.y;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;

  int ii,jj;
  ii=x+d_ijk[n].di; jj=y+d_ijk[n].dj;
  if(ii>0 && ii<d_sys.cimax-1 && jj>0 && jj<d_sys.cjmax-1){
    u_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
      =u[ii+d_sys.cimax*jj+n*d_sys.cimax*d_sys.cjmax];
    p_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
      =p[ii+d_sys.cimax*jj+n*d_sys.cimax*d_sys.cjmax];
  }
  else{
    u_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]=0.0f;
    p_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]=0.0f;
  }
}


__global__
void sumxy_Kernel(float* pf,float* pfx,float* pfy,int n){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;

  pfx[x+d_sys.cimax*y]=h(pf[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax])*(x+0.5f);
  pfy[x+d_sys.cimax*y]=h(pf[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax])*(y+0.5f);
}

__global__
void h_Kernel(float* pf_h,float* pf){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cimax;
  pf_h[x+d_sys.cimax*y]=h(pf[x+d_sys.cimax*y]);
}

__global__
void hu_Kernel(float* pf_h,float* pf){
  int n = blockIdx.y;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cimax;
  pf_h[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]=h(pf[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]);
}


__global__
void hv_Kernel(float* pf_h,float* pf){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cimax;
  pf_h[x+d_sys.cimax*y]=h(pf[x+d_sys.cimax*y])*d_sys.dx*d_sys.dy;
}


__global__
void reduce0(float *idata,float *odata, unsigned int n){

  extern __shared__ float sdata[];

  unsigned int tid = threadIdx.x; //1024
  unsigned int i = blockIdx.x * blockDim.x  + threadIdx.x; //(j*1024*2)+i(1024)

  sdata[tid] = (i < n) ? idata[i] : 0.0f;
  //sdata[tid] = (i < n) ? 0.1f : 0.0f;
  __syncthreads();
  //printf("%8.4f",sdata[tid]);

  for (unsigned int s=1; s<blockDim.x; s*=2) {
    if (tid % (2*s)==0) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }
  if (tid == 0) odata[blockIdx.x]  = sdata[0];
}

__global__
void reduce3(float *idata,float *odata, unsigned int n){

  extern __shared__ float sdata[];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * (blockDim.x*2)  + threadIdx.x;

  float mysum = (i < n) ? idata[i] : 0.0f;
  if (i + blockDim.x < n) mysum += idata[i + blockDim.x];
  sdata[tid] = mysum;
  __syncthreads();

  for (unsigned int s=1; s<blockDim.x; s*=2) {
    if (tid % (2*s)==0) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }
  if (tid == 0) odata[blockIdx.x]  = sdata[0];
}

__global__
void reduce3_i(int *idata,int *odata, unsigned int n){

  extern __shared__ int sdata_i[];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * (blockDim.x*2)  + threadIdx.x;

  int mysum = (i < n) ? idata[i] : 0;
  if (i + blockDim.x < n) mysum += idata[i + blockDim.x];
  sdata_i[tid] = mysum;
  __syncthreads();

  for (unsigned int s=1; s<blockDim.x; s*=2) {
    if (tid % (2*s)==0) {
      sdata_i[tid] += sdata_i[tid + s];
    }
    __syncthreads();
  }
  if (tid == 0) odata[blockIdx.x]  = sdata_i[0];
}




__global__
void init_all_Kernel(float* v){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.imax;
  int x = (index / d_sys.imax) % d_sys.jmax;

  v[x+d_sys.jmax*y]=0.0f;
}

__global__
void init_all_uphi_Kernel(float* u,float* phi){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.imax;
  int x = (index / d_sys.imax) % d_sys.jmax;

  u[x+d_sys.jmax*y]=0.0f;
  phi[x+d_sys.jmax*y]=0.0f;
}



__global__
void phi_Kernel(float* phi,const float* pf,const int n){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;

  phi[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]+=h(pf[x+d_sys.cjmax*y+n*d_sys.cimax*d_sys.cjmax]);
}

__global__
void u_m1_Kernel(float* u_n,float* hd_u_n,int n){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;

  u_n[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]=hd_u_n[x+d_sys.cimax*y];

}

__global__
void e_eta_Kernel(float* e_eta,float* pf,float* pf_n,int m){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;

  if(x>0 && x<d_sys.cimax-1 && y>0 && y<d_sys.cjmax-1){

    e_eta[(x+d_cells[m].cimin)+d_sys.jmax*(y+d_cells[m].cjmin)]
      +=(
	 (h(pf[(x+1)+d_sys.cimax*y+m*d_sys.cimax*d_sys.cjmax])
	  -h(pf[(x-1)+d_sys.cimax*y+m*d_sys.cimax*d_sys.cjmax]))*0.5f/d_sys.dx
	 *(h(pf_n[((x+1)+d_cells[m].cimin)+d_sys.jmax*(y+d_cells[m].cjmin)])
	   -h(pf_n[((x-1)+d_cells[m].cimin)+d_sys.jmax*(y+d_cells[m].cjmin)]))*0.5f/d_sys.dx
	 +(h(pf[x+d_sys.cimax*(y+1)+m*d_sys.cimax*d_sys.cjmax])
	   -h(pf[x+d_sys.cimax*(y-1)+m*d_sys.cimax*d_sys.cjmax]))*0.5f/d_sys.dy
	 *(h(pf_n[(x+d_cells[m].cimin)+d_sys.jmax*((y+1)+d_cells[m].cjmin)])
	   -h(pf_n[(x+d_cells[m].cimin)+d_sys.jmax*((y-1)+d_cells[m].cjmin)]))*0.5f/d_sys.dy
	 )*d_para.eta/6.0f
    ;

  }
}


__global__
void all_Kernel(float* uall,const float* pf,const int n){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;

  uall[(x+d_cells[n].cimin)+d_sys.jmax*(y+d_cells[n].cjmin)]+=pf[x+d_sys.cjmax*y+n*d_sys.cimax*d_sys.cjmax];
}


__global__
void all_uphi_Kernel(float* uall,float* phi,const float* u,int n){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % d_sys.cimax;
  int y = (index / d_sys.cimax) % d_sys.cjmax;

  uall[(x+d_cells[n].cimin)+d_sys.jmax*(y+d_cells[n].cjmin)]
    +=u[x+d_sys.cjmax*y+n*d_sys.cimax*d_sys.cjmax];
  phi[(x+d_cells[n].cimin)+d_sys.jmax*(y+d_cells[n].cjmin)]
    +=h(u[x+d_sys.cjmax*y+n*d_sys.cimax*d_sys.cjmax]);
}

__global__
void all_up_Kernel(float* uall,float* pall,
		      const float* u,const float* p,int n){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;

  uall[(x+d_cells[n].cimin)+d_sys.jmax*(y+d_cells[n].cjmin)]
    +=u[x+d_sys.cjmax*y+n*d_sys.cimax*d_sys.cjmax];
  pall[(x+d_cells[n].cimin)+d_sys.jmax*(y+d_cells[n].cjmin)]
    +=p[x+d_sys.cjmax*y+n*d_sys.cimax*d_sys.cjmax];
}


__global__
void init_normal_vector_field_Kernel(float* vx,float* vy){
//void init_normal_vector_field_Kernel(float** vx,float** vy){
//void init_normal_vector_field_Kernel(float** vx,float** vy,float** vz){
  int n = blockIdx.y;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;

  vx[n*d_sys.cimax*d_sys.cimax+x+d_sys.cimax*y]=0.0f;
  vy[n*d_sys.cimax*d_sys.cimax+x+d_sys.cimax*y]=0.0f;
  //vz[n][x+d_sys.cimax*y]=0.0f;
}

__global__
void init_normal_vector_field_all_Kernel(float* vx,float* vy){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.imax;
  int x = (index / d_sys.imax) % d_sys.jmax;

  vx[x+d_sys.imax*y]=0.0f;
  vy[x+d_sys.imax*y]=0.0f;
  //vz[n][x+d_sys.imax*y]=0.0f;
}

__global__
void normal_vector_field_Kernel(float* vx,float* vy,float* u){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % d_sys.cimax;
  int y = (index / d_sys.cimax) % d_sys.cjmax;
  int n = ((index / d_sys.cimax) / d_sys.cjmax) % d_cnum;

  vx[n*d_sys.cimax*d_sys.cimax+x+d_sys.cimax*y]=0.0f;
  vy[n*d_sys.cimax*d_sys.cimax+x+d_sys.cimax*y]=0.0f;

  if(x>0 && x<d_sys.cimax-1 && y>0 && y<d_sys.cimax-1){
    vx[n*d_sys.cimax*d_sys.cimax+x+d_sys.cimax*y]
      =(u[(x+1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cimax]
	-u[(x-1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cimax])*0.5f/d_sys.dx;
    vy[n*d_sys.cimax*d_sys.cimax+x+d_sys.cimax*y]
      =(u[x+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cimax]
	-u[x+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cimax])*0.5f/d_sys.dy;
    //vz[n][x+d_sys.cimax*y]=(u[n][x+d_sys.cimax*(y+1)]-u[n][x+d_sys.cimax*(y-1)])*0.50f/d_sys.dz;
  }

  float vabs=
    __fsqrt_rn(
	       vx[n*d_sys.cimax*d_sys.cimax+x+d_sys.cimax*y]
	       *vx[n*d_sys.cimax*d_sys.cimax+x+d_sys.cimax*y]
	       +vy[n*d_sys.cimax*d_sys.cimax+x+d_sys.cimax*y]
	       *vy[n*d_sys.cimax*d_sys.cimax+x+d_sys.cimax*y]
	       //+vz[n][x+d_sys.cimax*y]*vz[n][x+d_sys.cimax*y]
	       );
  if(vabs>0.00001f){
    vx[n*d_sys.cimax*d_sys.cimax+x+d_sys.cimax*y]/=vabs;
    vy[n*d_sys.cimax*d_sys.cimax+x+d_sys.cimax*y]/=vabs;
    //vz[n][x+d_sys.cimax*y]/=vabs;
  }
  else{
    vx[n*d_sys.cimax*d_sys.cimax+x+d_sys.cimax*y]=0.0f;
    vy[n*d_sys.cimax*d_sys.cimax+x+d_sys.cimax*y]=0.0f;
    //vz[n][x+d_sys.cimax*y]=0.0f;
  }
}

__global__
void normal_vector_field_all_Kernel(float* vx,float* vy,//float* vz,
				    float* pf){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.imax;
  int x = (index / d_sys.imax) % d_sys.jmax;

  vx[x+d_sys.imax*y]=0.0f;
  vy[x+d_sys.imax*y]=0.0f;

  if(x>0 && x<d_sys.imax-1 && y>0 && y<d_sys.jmax-1){
    vx[x+d_sys.imax*y]=(pf[(x+1)+d_sys.imax*y]-pf[(x-1)+d_sys.imax*y])*0.50f/d_sys.dx;
    vy[x+d_sys.imax*y]=(pf[x+d_sys.imax*(y+1)]-pf[x+d_sys.imax*(y-1)])*0.50f/d_sys.dy;
    //vz[n][x+d_sys.imax*y]=(pf[x+d_sys.imax*(y+1)]-pf[x+d_sys.imax*(y-1)])*0.50f/d_sys.dz;
  }

  float vabs=
    sqrtf(vx[x+d_sys.imax*y]*vx[x+d_sys.imax*y]+vy[x+d_sys.imax*y]*vy[x+d_sys.imax*y]
	  //+vz[x+d_sys.imax*y]*vz[x+d_sys.imax*y]
	  );
  if(vabs>0.0000f){
    vx[x+d_sys.imax*y]/=vabs;
    vy[x+d_sys.imax*y]/=vabs;
    //vz[x+d_sys.imax*y]/=vabs;
  }
  else{
    vx[x+d_sys.imax*y]=0.0f;
    vy[x+d_sys.imax*y]=0.0f;
    //vz[x+d_sys.imax*y]=0.0f;
  }
}


__global__
void normal_vector_field_all_2_Kernel(float* vx,float* vy,float* pf,int imin,int jmin,int imax,int jmax){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % imax;
  int y = (index / imax) % jmax;

  vx[(x+imin)+d_sys.imax*(y+jmin)]=0.0f;
  vy[(x+imin)+d_sys.imax*(y+jmin)]=0.0f;

  if(x+imin>0 && x+imin<d_sys.imax-1 && y+jmin>0 && y+jmin<d_sys.jmax-1){
    vx[(x+imin)+d_sys.imax*(y+jmin)]=(pf[(x+imin+1)+d_sys.imax*(y+jmin)]
				      -pf[(x+imin-1)+d_sys.imax*(y+jmin)])*0.5f/d_sys.dx;
    vy[(x+imin)+d_sys.imax*(y+jmin)]=(pf[(x+imin)+d_sys.imax*(y+jmin+1)]
				      -pf[(x+imin)+d_sys.imax*(y+jmin-1)])*0.5f/d_sys.dy;
  }

  float vabs=
    sqrtf(vx[(x+imin)+d_sys.imax*(y+jmin)]*vx[(x+imin)+d_sys.imax*(y+jmin)]
	  +vy[(x+imin)+d_sys.imax*(y+jmin)]*vy[(x+imin)+d_sys.imax*(y+jmin)]
	  //+vz[x+d_sys.imax*y]*vz[x+d_sys.imax*y]
	  );
  if(vabs>0.0000f){
    vx[(x+imin)+d_sys.imax*(y+jmin)]/=vabs;
    vy[(x+imin)+d_sys.imax*(y+jmin)]/=vabs;
    //vz[x+d_sys.imax*y]/=vabs;
  }
  else{
    vx[(x+imin)+d_sys.imax*(y+jmin)]=0.0f;
    vy[(x+imin)+d_sys.imax*(y+jmin)]=0.0f;
    //vz[x+d_sys.imax*y]=0.0f;
  }
}



__global__
void normal_vector_field_sc_Kernel(float* vx_s,float* vy_s,float* s,
				   float* vx_c,float* vy_c,float* c){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.imax;
  int x = (index / d_sys.imax) % d_sys.jmax;

  vx_s[x+d_sys.imax*y]=0.0f; vy_s[x+d_sys.imax*y]=0.0f;
  vx_c[x+d_sys.imax*y]=0.0f; vy_c[x+d_sys.imax*y]=0.0f;

  if(x>0 && x<d_sys.imax-1 && y>0 && y<d_sys.jmax-1){
    vx_s[x+d_sys.imax*y]=(s[(x+1)+d_sys.imax*y]-s[(x-1)+d_sys.imax*y])*0.50f/d_sys.dx;
    vy_s[x+d_sys.imax*y]=(s[x+d_sys.imax*(y+1)]-s[x+d_sys.imax*(y-1)])*0.50f/d_sys.dy;
    vx_c[x+d_sys.imax*y]=(c[(x+1)+d_sys.imax*y]-c[(x-1)+d_sys.imax*y])*0.50f/d_sys.dx;
    vy_c[x+d_sys.imax*y]=(c[x+d_sys.imax*(y+1)]-c[x+d_sys.imax*(y-1)])*0.50f/d_sys.dy;
    //vz[n][x+d_sys.imax*y]=(pf[x+d_sys.imax*(y+1)]-pf[x+d_sys.imax*(y-1)])*0.50f/d_sys.dz;
  }

  float vabs_s=sqrtf(vx_s[x+d_sys.imax*y]*vx_s[x+d_sys.imax*y]
		     +vy_s[x+d_sys.imax*y]*vy_s[x+d_sys.imax*y]
		     //+vz[x+d_sys.imax*y]*vz[x+d_sys.imax*y]
		     );
  float vabs_c=sqrtf(vx_c[x+d_sys.imax*y]*vx_c[x+d_sys.imax*y]
		     +vy_c[x+d_sys.imax*y]*vy_c[x+d_sys.imax*y]
		     //+vz[x+d_sys.imax*y]*vz[x+d_sys.imax*y]
		     );

  if(vabs_s>0.0000f){
    vx_s[x+d_sys.imax*y]/=vabs_s; vy_s[x+d_sys.imax*y]/=vabs_s; //vz[x+d_sys.imax*y]/=vabs;
  }
  else{
    vx_s[x+d_sys.imax*y]=0.0f; vy_s[x+d_sys.imax*y]=0.0f; //vz[x+d_sys.imax*y]=0.0f;
  }

  if(vabs_c>0.0000f){
    vx_c[x+d_sys.imax*y]/=vabs_c; vy_c[x+d_sys.imax*y]/=vabs_c; //vz[x+d_sys.imax*y]/=vabs;
  }
  else{
    vx_c[x+d_sys.imax*y]=0.0f; vy_c[x+d_sys.imax*y]=0.0f; //vz[x+d_sys.imax*y]=0.0f;
  }
}

__global__
void normal_vector_field_sc_2_Kernel(float* vx_s,float* vy_s,float* s,
				     float* vx_c,float* vy_c,float* c,
				     int imin,int jmin,int imax,int jmax){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % imax;
  int y = (index / imax) % jmax;

  vx_s[(x+imin)+d_sys.imax*(y+jmin)]=0.0f; vy_s[(x+imin)+d_sys.imax*(y+jmin)]=0.0f;
  vx_c[(x+imin)+d_sys.imax*(y+jmin)]=0.0f; vy_c[(x+imin)+d_sys.imax*(y+jmin)]=0.0f;

  if(x+imin>0 && x+imin<d_sys.imax-1 && y+jmin>0 && y+jmin<d_sys.jmax-1){
    vx_s[(x+imin)+d_sys.imax*(y+jmin)]=(s[(x+imin+1)+d_sys.imax*(y+jmin)]
					-s[(x+imin-1)+d_sys.imax*(y+jmin)])*0.5f/d_sys.dx;
    vy_s[(x+imin)+d_sys.imax*(y+jmin)]=(s[(x+imin)+d_sys.imax*(y+jmin+1)]
					-s[(x+imin)+d_sys.imax*(y+jmin-1)])*0.5f/d_sys.dy;
    vx_c[(x+imin)+d_sys.imax*(y+jmin)]=(c[(x+imin+1)+d_sys.imax*(y+jmin)]
					-c[(x+imin-1)+d_sys.imax*(y+jmin)])*0.5f/d_sys.dx;
    vy_c[(x+imin)+d_sys.imax*(y+jmin)]=(c[(x+imin)+d_sys.imax*(y+jmin+1)]
					-c[(x+imin)+d_sys.imax*(y+jmin-1)])*0.5f/d_sys.dy;
    //vz[n][x+d_sys.imax*y]=(pf[x+d_sys.imax*(y+1)]-pf[x+d_sys.imax*(y-1)])*0.50f/d_sys.dz;
  }

  float vabs_s=sqrtf(vx_s[(x+imin)+d_sys.imax*(y+jmin)]*vx_s[(x+imin)+d_sys.imax*(y+jmin)]
		     +vy_s[(x+imin)+d_sys.imax*(y+jmin)]*vy_s[(x+imin)+d_sys.imax*(y+jmin)]
		     //+vz[x+d_sys.imax*y]*vz[x+d_sys.imax*y]
		     );
  float vabs_c=sqrtf(vx_c[(x+imin)+d_sys.imax*(y+jmin)]*vx_c[(x+imin)+d_sys.imax*(y+jmin)]
		     +vy_c[(x+imin)+d_sys.imax*(y+jmin)]*vy_c[(x+imin)+d_sys.imax*(y+jmin)]
		     //+vz[x+d_sys.imax*y]*vz[x+d_sys.imax*y]
		     );

  if(vabs_s>0.0000f){
    vx_s[(x+imin)+d_sys.imax*(y+jmin)]/=vabs_s;
    vy_s[(x+imin)+d_sys.imax*(y+jmin)]/=vabs_s; //vz[x+d_sys.imax*y]/=vabs;
  }
  else{
    vx_s[(x+imin)+d_sys.imax*(y+jmin)]=0.0f;
    vy_s[(x+imin)+d_sys.imax*(y+jmin)]=0.0f; //vz[x+d_sys.imax*y]=0.0f;
  }

  if(vabs_c>0.0000f){
    vx_c[(x+imin)+d_sys.imax*(y+jmin)]/=vabs_c;
    vy_c[(x+imin)+d_sys.imax*(y+jmin)]/=vabs_c; //vz[x+d_sys.imax*y]/=vabs;
  }
  else{
    vx_c[(x+imin)+d_sys.imax*(y+jmin)]=0.0f;
    vy_c[(x+imin)+d_sys.imax*(y+jmin)]=0.0f; //vz[x+d_sys.imax*y]=0.0f;
  }
}




__global__
void boundary_normal_vector_field_Kernel(float* vx,float* vy
//void boundary_normal_vector_field_Kernel(float** vx,float** vy
					 //,float** vz
					 ){
  int n = blockIdx.y;
  int y = blockIdx.x * blockDim.x + threadIdx.x;

  vx[n*d_sys.cimax*d_sys.cimax+0+d_sys.cimax*y]
    =vx[n*d_sys.cimax*d_sys.cimax+(d_sys.cimax-2)+d_sys.cimax*y];
  vy[n*d_sys.cimax*d_sys.cimax+0+d_sys.cimax*y]
    =vy[n*d_sys.cimax*d_sys.cimax+(d_sys.cimax-2)+d_sys.cimax*y];
  // vz[n][x+d_sys.cimax*y]=vz[n][(d_sys.cimax-2)+d_sys.cimax*y];
  vx[n*d_sys.cimax*d_sys.cimax+y+d_sys.cimax*0]
    =vx[n*d_sys.cimax*d_sys.cimax+y+d_sys.cimax*(d_sys.cimax-2)];
  vy[n*d_sys.cimax*d_sys.cimax+y+d_sys.cimax*0]
    =vy[n*d_sys.cimax*d_sys.cimax+y+d_sys.cimax*(d_sys.cimax-2)];
  //vz[n][x+d_sys.cimax*y]=vz[n][x+d_sys.cimax*(d_sys.cjmax-2)];

  vx[n*d_sys.cimax*d_sys.cimax+d_sys.cimax-1+d_sys.cimax*y]
    =vx[n*d_sys.cimax*d_sys.cimax+(d_sys.cimax-2)+d_sys.cimax*y];
  vy[n*d_sys.cimax*d_sys.cimax+d_sys.cimax-1+d_sys.cimax*y]
    =vy[n*d_sys.cimax*d_sys.cimax+(d_sys.cimax-2)+d_sys.cimax*y];
  // vz[n][x+d_sys.cimax*y]=vz[n][(d_sys.cimax-2)+d_sys.cimax*y];
  vx[n*d_sys.cimax*d_sys.cimax+y+d_sys.cimax*(d_sys.cimax-1)]
    =vx[n*d_sys.cimax*d_sys.cimax+y+d_sys.cimax*1];
  vy[n*d_sys.cimax*d_sys.cimax+y+d_sys.cimax*(d_sys.cimax-1)]
    =vy[n*d_sys.cimax*d_sys.cimax+y+d_sys.cimax*1];
  //vz[n][x+d_sys.cimax*y]=vz[n][x+d_sys.cimax*(d_sys.cjmax-2)];

}

__global__
void boundary_normal_vector_field_all_Kernel(float* vx,float* vy
//void boundary_normal_vector_field_Kernel(float** vx,float** vy
					 //,float** vz
					 ){
  //int n = blockIdx.y;
  int y = blockIdx.x * blockDim.x + threadIdx.x;

  vx[0+d_sys.imax*y]=vx[(d_sys.imax-2)+d_sys.imax*y];
  vy[0+d_sys.imax*y]=vy[(d_sys.imax-2)+d_sys.imax*y];
  // vz[n][x+d_sys.imax*y]=vz[n][(d_sys.imax-2)+d_sys.imax*y];
  vx[y+d_sys.imax*0]=vx[y+d_sys.imax*(d_sys.imax-2)];
  vy[y+d_sys.imax*0]=vy[y+d_sys.imax*(d_sys.imax-2)];
  //vz[n][x+d_sys.imax*y]=vz[n][x+d_sys.imax*(d_sys.jmax-2)];

  vx[d_sys.imax-1+d_sys.imax*y]=vx[(d_sys.imax-2)+d_sys.imax*y];
  vy[d_sys.imax-1+d_sys.imax*y]=vy[(d_sys.imax-2)+d_sys.imax*y];
  // vz[n][x+d_sys.imax*y]=vz[n][(d_sys.imax-2)+d_sys.imax*y];
  vx[y+d_sys.imax*(d_sys.imax-1)]=vx[y+d_sys.imax*1];
  vy[y+d_sys.imax*(d_sys.imax-1)]=vy[y+d_sys.imax*1];
  //vz[n][x+d_sys.imax*y]=vz[n][x+d_sys.imax*(d_sys.jmax-2)];
}

__global__
void boundary_normal_vector_field_sc_Kernel(float* vx_s,float* vy_s, //float* vz,
					     float* vx_c,float* vy_c //float* vz,
					     ){
  int y = blockIdx.x * blockDim.x + threadIdx.x;

  vx_s[0+d_sys.imax*y]=vx_s[(d_sys.imax-2)+d_sys.imax*y];
  vy_s[0+d_sys.imax*y]=vy_s[(d_sys.imax-2)+d_sys.imax*y];
  // vz_s[n][x+d_sys.imax*y]=vz_s[n][(d_sys.imax-2)+d_sys.imax*y];
  vx_s[y+d_sys.imax*0]=vx_s[y+d_sys.imax*(d_sys.imax-2)];
  vy_s[y+d_sys.imax*0]=vy_s[y+d_sys.imax*(d_sys.imax-2)];
  //vz_s[n][x+d_sys.imax*y]=vz_s[n][x+d_sys.imax*(d_sys.jmax-2)];

  vx_s[d_sys.imax-1+d_sys.imax*y]=vx_s[(d_sys.imax-2)+d_sys.imax*y];
  vy_s[d_sys.imax-1+d_sys.imax*y]=vy_s[(d_sys.imax-2)+d_sys.imax*y];
  // vz_s[n][x+d_sys.imax*y]=vz_s[n][(d_sys.imax-2)+d_sys.imax*y];
  vx_s[y+d_sys.imax*(d_sys.imax-1)]=vx_s[y+d_sys.imax*1];
  vy_s[y+d_sys.imax*(d_sys.imax-1)]=vy_s[y+d_sys.imax*1];
  //vz_s[n][x+d_sys.imax*y]=vz_s[n][x+d_sys.imax*(d_sys.jmax-2)];

  vx_c[0+d_sys.imax*y]=vx_c[(d_sys.imax-2)+d_sys.imax*y];
  vy_c[0+d_sys.imax*y]=vy_c[(d_sys.imax-2)+d_sys.imax*y];
  // vz_c[n][x+d_sys.imax*y]=vz_c[n][(d_sys.imax-2)+d_sys.imax*y];
  vx_c[y+d_sys.imax*0]=vx_c[y+d_sys.imax*(d_sys.imax-2)];
  vy_c[y+d_sys.imax*0]=vy_c[y+d_sys.imax*(d_sys.imax-2)];
  //vz_c[n][x+d_sys.imax*y]=vz_c[n][x+d_sys.imax*(d_sys.jmax-2)];

  vx_c[d_sys.imax-1+d_sys.imax*y]=vx_c[(d_sys.imax-2)+d_sys.imax*y];
  vy_c[d_sys.imax-1+d_sys.imax*y]=vy_c[(d_sys.imax-2)+d_sys.imax*y];
  // vz_c[n][x+d_sys.imax*y]=vz_c[n][(d_sys.imax-2)+d_sys.imax*y];
  vx_c[y+d_sys.imax*(d_sys.imax-1)]=vx_c[y+d_sys.imax*1];
  vy_c[y+d_sys.imax*(d_sys.imax-1)]=vy_c[y+d_sys.imax*1];
  //vz_c[n][x+d_sys.imax*y]=vz_c[n][x+d_sys.imax*(d_sys.jmax-2)];

}



__global__
void reshaping_Kernel(float* vx,float* vy, //const float** vz,
 		      float* u,float* u_temp,float dtau){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % d_sys.cimax;
  int y = (index / d_sys.cimax) % d_sys.cjmax;
  int n = ((index / d_sys.cimax) / d_sys.cjmax) % d_cnum;

  if(x>0 && x<d_sys.cimax-1 && y>0 && y<d_sys.cimax-1){
    u_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cimax]
      =u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cimax]
      +dtau*d_para.D_u*laplacian8(u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax],
				  u[(x-1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax],
				  u[(x+1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax],
				  u[x+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cjmax],
				  u[x+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cjmax],
				  u[(x-1)+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cjmax],
				  u[(x+1)+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cjmax],
				  u[(x-1)+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cjmax],
				  u[(x+1)+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cjmax])
      // +dtau*d_para.D_u*laplacian4(u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cimax],
      // 	     			  u[(x-1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cimax],
      // 				  u[(x+1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cimax],
      // 	     			  u[x+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cimax],
      // 				  u[x+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cimax])
      -dtau*sqrtf(2.0f*d_para.D_u)*
      (
       (u[(x+1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cimax]
	*(1.0f-u[(x+1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cimax])
	*vx[(x+1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cimax]
	-u[(x-1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cimax]
	*(1.0f-u[(x-1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cimax])
	*vx[(x-1)+d_sys.cimax*y+n*d_sys.cimax*d_sys.cimax])
       *0.5f/d_sys.dx
       +(u[x+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cimax]
	 *(1.0f-u[x+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cimax])
	 *vy[x+d_sys.cimax*(y+1)+n*d_sys.cimax*d_sys.cimax]
	 -u[x+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cimax]
	 *(1.0f-u[x+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cimax])
	 *vy[x+d_sys.cimax*(y-1)+n*d_sys.cimax*d_sys.cimax])
       *0.5f/d_sys.dy
       )
      ;
  }
}


__global__
void reshaping_all_Kernel(float* vx,float* vy, //const float** vz,
			  float* pf,float* pf_temp,float dtau,float D){
// void reshaping_Kernel(float** vx,float** vy, //const float** vz,
// 		      float** u,float** u_temp,float dtau){
  //int n = blockIdx.y;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.imax;
  int x = (index / d_sys.imax) % d_sys.jmax;

  if(x>0 && x<d_sys.imax-1 && y>0 && y<d_sys.imax-1){
    pf_temp[x+d_sys.imax*y]
      =pf[x+d_sys.imax*y]
      +dtau*D
      //*laplacian4(i,j)
      *(
	(pf[x+d_sys.imax*(y-1)]-2.00f*pf[x+d_sys.imax*y]+pf[x+d_sys.imax*(y+1)])/(d_sys.dx*d_sys.dx)
      	+(pf[(x-1)+d_sys.imax*y]-2.00f*pf[x+d_sys.imax*y]+pf[(x+1)+d_sys.imax*y])/(d_sys.dy*d_sys.dy)
	)
      // *(
      // 	(pf[x+d_sys.imax*(y-1)]-2.00f*pf[x+d_sys.imax*y]+pf[x+d_sys.imax*(y+1)])/(d_sys.dx*d_sys.dx)
      // 	+(pf[(x-1)+d_sys.imax*y]-2.00f*pf[x+d_sys.imax*y]+pf[(x+1)+d_sys.imax*y])/(d_sys.dy*d_sys.dy)
      // 	+(0.50f*(pf[(x-1)+d_sys.imax*(y-1)]+pf[(x-1)+d_sys.imax*(y+1)]
      // 		 +pf[(x+1)+d_sys.imax*(y-1)]+pf[(x+1)+d_sys.imax*(y+1)])
      // 	  -2.0f*pf[x+d_sys.imax*y])/(d_sys.dx*d_sys.dy)
      // 	)*0.50f
      -dtau*sqrtf(2.0f*D)*
      (
       (pf[(x+1)+d_sys.imax*y]*(1.0f-pf[(x+1)+d_sys.imax*y])*vx[(x+1)+d_sys.imax*y]
	-pf[(x-1)+d_sys.imax*y]*(1.0f-pf[(x-1)+d_sys.imax*y])*vx[(x-1)+d_sys.imax*y])
       *0.50f/d_sys.dx
       +(pf[x+d_sys.imax*(y+1)]*(1.0f-pf[x+d_sys.imax*(y+1)])*vy[x+d_sys.imax*(y+1)]
	 -pf[x+d_sys.imax*(y-1)]*(1.0f-pf[x+d_sys.imax*(y-1)])*vy[x+d_sys.imax*(y-1)])
       *0.50f/d_sys.dy
       )
      ;
  }
}

__global__
void reshaping_all_2_Kernel(float* vx,float* vy, //const float** vz,
			    float* pf,float* pf_temp,float dtau,float D,
			    int imin,int jmin,int imax,int jmax){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % imax;
  int y = (index / imax) % jmax;

  if(x+imin>0 && x+imin<d_sys.imax-1 && y+jmin>0 && y+jmin<d_sys.imax-1){
    pf_temp[(x+imin)+d_sys.imax*(y+jmin)]
      =pf[(x+imin)+d_sys.imax*(y+jmin)]
      +dtau*D
      *laplacian8(pf[(x+imin)+d_sys.imax*(y+jmin)],
		  pf[(x+imin-1)+d_sys.imax*(y+jmin)],
		  pf[(x+imin+1)+d_sys.imax*(y+jmin)],
		  pf[(x+imin)+d_sys.imax*(y+jmin-1)],
		  pf[(x+imin)+d_sys.imax*(y+jmin+1)],
		  pf[(x+imin-1)+d_sys.imax*(y+jmin-1)],
		  pf[(x+imin+1)+d_sys.imax*(y+jmin-1)],
		  pf[(x+imin-1)+d_sys.imax*(y+jmin+1)],
		  pf[(x+imin+1)+d_sys.imax*(y+jmin+1)])
      // //*laplacian4(i,j)
      // +dtau*D
      // *(
      // 	(pf[(x+imin-1)+d_sys.imax*(y+jmin)]
      // 	 -2.0f*pf[(x+imin)+d_sys.imax*(y+jmin)]
      // 	 +pf[(x+imin+1)+d_sys.imax*(y+jmin)])/(d_sys.dx*d_sys.dx)
      // 	+(pf[(x+imin)+d_sys.imax*(y+jmin-1)]
      // 	  -2.0f*pf[(x+imin)+d_sys.imax*(y+jmin)]
      // 	  +pf[(x+imin)+d_sys.imax*(y+jmin+1)])/(d_sys.dy*d_sys.dy)
      // 	)
      -dtau*sqrtf(2.0f*D)*
      (
       (pf[(x+imin+1)+d_sys.imax*(y+jmin)]*(1.0f-pf[(x+imin+1)+d_sys.imax*(y+jmin)])
	*vx[(x+imin+1)+d_sys.imax*(y+jmin)]
	-pf[(x+imin-1)+d_sys.imax*(y+jmin)]*(1.0f-pf[(x+imin-1)+d_sys.imax*(y+jmin)])
	*vx[(x+imin-1)+d_sys.imax*(y+jmin)])
       *0.5f/d_sys.dx
       +(pf[(x+imin)+d_sys.imax*(y+jmin+1)]*(1.0f-pf[(x+imin)+d_sys.imax*(y+jmin+1)])
	 *vy[(x+imin)+d_sys.imax*(y+jmin+1)]
	 -pf[(x+imin)+d_sys.imax*(y+jmin-1)]*(1.0f-pf[(x+imin)+d_sys.imax*(y+jmin-1)])
	 *vy[(x+imin)+d_sys.imax*(y+jmin-1)])
       *0.5f/d_sys.dy
       )
      ;
  }
}



__global__
void u_temp_Kernel(float* u,float** u_temp){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;

  u[x+d_sys.cimax*y]=u_temp[0][x+d_sys.cimax*y];
}

__global__
void dev_Kernel(float*dev,float* u,float* u_temp){

  int n = blockIdx.y;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;

  dev[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cimax]
   =fabsf(u_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cimax]
	  -u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cimax]);
}

__global__
void dev2_Kernel(float*dev,float* u,float* u_temp){

  int n = blockIdx.y;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;

  dev[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]
    =u_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cimax]
    -u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax];
}


__global__
void dev_all_Kernel(float*dev,float* pf,float* pf_temp){

  //int n = blockIdx.y;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.imax;
  int x = (index / d_sys.imax) % d_sys.jmax;

  //  dev[x+d_sys.imax*y]
  dev[x+d_sys.imax*y]=fabsf(pf_temp[x+d_sys.imax*y]-pf[x+d_sys.imax*y]);
}

__global__
void dev_all_2_Kernel(float*dev,float* pf,float* pf_temp,int imin,int jmin,int imax,int jmax){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % imax;
  int y = (index / imax) % jmax;

  dev[(x+imin)+d_sys.imax*(y+jmin)]=fabsf(pf_temp[(x+imin)+d_sys.imax*(y+jmin)]-pf[(x+imin)+d_sys.imax*(y+jmin)]);
}


__global__
void dev_i_all_Kernel(int*dev_i,float* pf,float* pf_temp,int fi){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.imax;
  int x = (index / d_sys.imax) % d_sys.jmax;

  dev_i[x+d_sys.imax*y]=(int)(fabsf(pf_temp[x+d_sys.imax*y]
				       -pf[x+d_sys.imax*y])*fi);
  dev_i[x+d_sys.imax*y]=dev_i[x+d_sys.imax*y]*fi;
}



__global__
void d_dev_Kernel(float*dev,float* dd_dev,int n){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;

  dd_dev[x+d_sys.cimax*y]=dev[n*d_sys.cimax*d_sys.cimax+x+d_sys.cimax*y];
}

__global__
void init_medium_Kernel(float*d_c,float*d_uall){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.imax;
  int x = (index / d_sys.imax) % d_sys.jmax;

  d_c[x+d_sys.jmax*y]=1.0f-d_uall[x+d_sys.jmax*y];
  if(d_c[x+d_sys.jmax*y]<0.0f) d_c[x+d_sys.jmax*y]=0.0f;
  //if(d_c[x+d_sys.jmax*y]>1.0f) d_c[x+d_sys.jmax*y]=1.0f;
}


__global__
void boundary_dirichlet_Kernel(float*v,float*v_temp){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.imax;

  v[0+d_sys.imax*y]=1.0f; v[y+d_sys.imax*0]=1.0f;
  v[d_sys.imax-1+d_sys.imax*y]=1.0f; v[y+d_sys.imax*(d_sys.jmax-1)]=1.0f;
  v_temp[0+d_sys.imax*y]=1.0f; v_temp[y+d_sys.imax*0]=1.0f;
  v_temp[d_sys.imax-1+d_sys.imax*y]=1.0f; v_temp[y+d_sys.imax*(d_sys.jmax-1)]=1.0f;
}


__device__
float distance(float r1x,float r1y,float r2x,float r2y){
  return sqrtf((r1x-r2x)*(r1x-r2x)+(r1y-r2y)*(r1y-r2y));
}


__global__
void copy_mother_cell_Kernel(float*u,float*m_u,int m){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;

  m_u[x+d_sys.cimax*y]=u[x+d_sys.cimax*y+m*d_sys.cimax*d_sys.cjmax];
}

__global__
void set_daughter_cells_Kernel(float*u,float*u_temp,
			       float r1x,float r1y, float r2x,float r2y,
			       int m,int d2){
  int n = blockIdx.y;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;

  if(n==m){
    float rx,ry;
    rx=d_cells[m].Gx-d_sys.cimax*0.5*d_sys.dx+(x+0.5)*d_sys.dx;
    ry=d_cells[m].Gy-d_sys.cjmax*0.5*d_sys.dy+(y+0.5)*d_sys.dy;
    //printf("%lf\t%lf\n",rx,ry);

    float d=distance(r1x,r1y,r2x,r2y);
    float g=(r1x-r2x)/d*(rx-(r1x+r2x)*0.5f)+(r1y-r2y)/d*(ry-(r1y+r2y)*0.5f);
    float chi=0.5f*(1+tanhf(g/d_para.ep_d));
    u_temp[x+d_sys.cimax*y+d2*d_sys.cimax*d_sys.cjmax]=u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]*(1-chi);
    u_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]=u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]*chi;
  }
  else{
    u_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]=u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax];
  }
}

__global__
void set_daughter_cells_2_Kernel(float*u,float*u_temp,float*p,float*p_temp,
				 float r1x,float r1y, float r2x,float r2y,
				 int m,int d2){
  int n = blockIdx.y;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;

  if(n==m){
    float rx,ry;
    rx=d_cells[m].Gx-d_sys.cimax*0.5*d_sys.dx+(x+0.5)*d_sys.dx;
    ry=d_cells[m].Gy-d_sys.cjmax*0.5*d_sys.dy+(y+0.5)*d_sys.dy;
    //printf("%lf\t%lf\n",rx,ry);

    float d=distance(r1x,r1y,r2x,r2y);
    float g=(r1x-r2x)/d*(rx-(r1x+r2x)*0.5f)+(r1y-r2y)/d*(ry-(r1y+r2y)*0.5f);
    float chi=0.5f*(1+tanhf(g/d_para.ep_d));
    u_temp[x+d_sys.cimax*y+d2*d_sys.cimax*d_sys.cjmax]=u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]*(1-chi);
    u_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]=u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]*chi;
    p_temp[x+d_sys.cimax*y+d2*d_sys.cimax*d_sys.cjmax]=p[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]*(1-chi);
    p_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]=p[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]*chi;
  }
  else{
    u_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]=u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax];
    p_temp[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]=p[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax];
  }
}


__global__
void force_field_Kernel(float*m_u,float*e_eta,
			float r1x,float r1y,float r2x,float r2y,
			float*f1x,float*f1y,
			float*f2x,float*f2y,
			int n){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;


  f1x[x+d_sys.cimax*y]=0.0f;
  f1y[x+d_sys.cimax*y]=0.0f;
  f2x[x+d_sys.cimax*y]=0.0f;
  f2y[x+d_sys.cimax*y]=0.0f;

  float rx,ry;
  rx=(float)(d_cells[n].Gx-d_sys.cimax*0.5f*d_sys.dx+(x+0.5f)*d_sys.dx);
  ry=(float)(d_cells[n].Gy-d_sys.cjmax*0.5f*d_sys.dy+(y+0.5f)*d_sys.dy);

  if(distance(r1x,r1y,rx,ry)<distance(r2x,r2y,rx,ry)){
    f1x[x+d_sys.cimax*y]=
      (d_para.rho0-d_para.rhoe*e_eta[(x+d_cells[n].cimin)+d_sys.jmax*(y+d_cells[n].cjmin)])
      *m_u[x+d_sys.cimax*y]*(1.0f-m_u[x+d_sys.cimax*y])*(rx-r1x);
    f1y[x+d_sys.cimax*y]=
      (d_para.rho0-d_para.rhoe*e_eta[(x+d_cells[n].cimin)+d_sys.jmax*(y+d_cells[n].cjmin)])
      *m_u[x+d_sys.cimax*y]*(1.0f-m_u[x+d_sys.cimax*y])*(ry-r1y);
  }
  if(distance(r1x,r1y,rx,ry)>distance(r2x,r2y,rx,ry)){
    f2x[x+d_sys.cimax*y]=
      (d_para.rho0-d_para.rhoe*e_eta[(x+d_cells[n].cimin)+d_sys.jmax*(y+d_cells[n].cjmin)])
      *m_u[x+d_sys.cimax*y]*(1.0f-m_u[x+d_sys.cimax*y])*(rx-r2x);
    f2y[x+d_sys.cimax*y]=
      (d_para.rho0-d_para.rhoe*e_eta[(x+d_cells[n].cimin)+d_sys.jmax*(y+d_cells[n].cjmin)])
      *m_u[x+d_sys.cimax*y]*(1.0f-m_u[x+d_sys.cimax*y])*(ry-r2y);
  }

  //printf("%lf\t%lf\n",f1x[x+d_sys.cimax*y],f1y[x+d_sys.cimax*y]);

}

__global__
void set_seed_p_Kernel(float*p,float Pcx,float Pcy,int n){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;


  float r
    =sqrtf((d_sys.dx*(x+d_cells[n].cimin+0.5f)-Pcx)*(d_sys.dx*(x+d_cells[n].cimin+0.5f)-Pcx)
	   +(d_sys.dy*(y+d_cells[n].cjmin+0.5f)-Pcy)*(d_sys.dy*(y+d_cells[n].cjmin+0.5f)-Pcy));

  //if(r<d_para.p_r) p[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]=1.0f;
  p[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]+=(1.0f-tanhf((r-d_para.p_r)/(sqrtf(2.0f*d_para.D_p))))*0.5f;
  if(p[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]>1.0f)
    p[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]=1.0f;
}

__global__
void set_seed_s_Kernel(float*u,float*s,float Pcx,float Pcy,int n){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % d_sys.cimax;
  int y = (index / d_sys.cimax) % d_sys.cjmax;


  float r
    =sqrtf((d_sys.dx*(x+d_cells[n].cimin+0.5f)-Pcx)*(d_sys.dx*(x+d_cells[n].cimin+0.5f)-Pcx)
	   +(d_sys.dy*(y+d_cells[n].cjmin+0.5f)-Pcy)*(d_sys.dy*(y+d_cells[n].cjmin+0.5f)-Pcy));

  float seed
    =(1.0f-tanhf((r-d_para.p_r)/(sqrtf(2.0f*d_para.D_u))))*0.5f;
  if(seed>0.00001f){
    s[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]+=seed;
    if(s[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]>1.0000f)
      s[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]=1.0f;
    u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]-=seed;
    if(u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]<0.0000001f)
      u[x+d_sys.cimax*y+n*d_sys.cimax*d_sys.cjmax]=0.0f;
  }

}

__global__
void overlap_us_Kernel(float *u_all,float *s,float *overlap){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.imax;
  int x = (index / d_sys.imax) % d_sys.jmax;

  overlap[x+d_sys.imax*y]=s[x+d_sys.imax*y]*(1.0f-s[x+d_sys.imax*y])*u_all[x+d_sys.imax*y]*d_sys.dx*d_sys.dy;
}


__global__
void debug_phi_Kernel(float* phim,float** u,int n){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int y = index % d_sys.cimax;
  int x = (index / d_sys.cimax) % d_sys.cjmax;
  phim[(x+d_cells[n].cimin)+d_sys.imax*(y+d_cells[n].cjmin)]-=h(u[n][x+d_sys.cimax*y]);
}


/////////////////////////////////////////////////////////////////////////////////
//
//  function code
//
/////////////////////////////////////////////////////////////////////////////////

float distance_h(float (&r1)[2],float (&r2)[2]){
  return sqrt((r1[0]-r2[0])*(r1[0]-r2[0])+(r1[1]-r2[1])*(r1[1]-r2[1]));
}


float volume_h(float*d_u){
  float result;

  int n=CIMAX*CJMAX;
  int th=WARP;
  //int blocks=(n-1)/th+1;  //reduce0
  int blocks=(n-1)/(2*th)+1;  //reduce3

  int shared_mem_size=2*th*sizeof(float);

  float *d1_out, *d2_out;
  hipMalloc((void**)&d1_out,sizeof(float)*blocks);
  hipMalloc((void**)&d2_out,sizeof(float)*blocks);

  float *dd_u;
  hipMalloc((void**)&dd_u,sizeof(float)*n);
  h_Kernel<<<n/th,th>>>(dd_u,d_u);

  float **in=&dd_u,**out=&d1_out;
  while (blocks>1) {
    //reduce0<<<blocks, th, shared_mem_size>>>(*in, *out, n);
    reduce3<<<blocks, th, shared_mem_size>>>(*in, *out, n);
    hipError_t err = hipGetLastError();
    if(err != hipSuccess){
      puts(hipGetErrorString(err));
    }
    if(*out==d1_out){
      out=&d2_out; in=&d1_out;
    }
    else{
      out=&d1_out; in=&d2_out;
    }
    n=blocks;
    // blocks=(blocks-1)/th+1;  //reduce0
    blocks=(blocks-1)/(2*th)+1;  //reduce3
    hipDeviceSynchronize();
  }
  //reduce0<<<blocks,th,shared_mem_size>>>(*in,*out,n);
  reduce3<<<blocks,th,shared_mem_size>>>(*in,*out,n);
  hipMemcpy(&result,*out,sizeof(float),hipMemcpyDeviceToHost);

  hipFree(d1_out);
  hipFree(d2_out);
  d1_out=NULL;
  d2_out=NULL;
  hipFree(dd_u);
  dd_u=NULL;

  return result*DX*DY;
  //return result;
}



float volume(float*d_u){
  float result;

  int n=CIMAX*CJMAX;
  int th=WARP;
  //int blocks=(n-1)/th+1;  //reduce0
  int blocks=(n-1)/(2*th)+1;  //reduce3

  int shared_mem_size=2*th*sizeof(float);

  float *d1_out, *d2_out;
  hipMalloc((void**)&d1_out,sizeof(float)*blocks);
  hipMalloc((void**)&d2_out,sizeof(float)*blocks);

  float **in=&d_u,**out=&d1_out;
  while (blocks>1) {
    //reduce0<<<blocks, th, shared_mem_size>>>(*in, *out, n);
    reduce3<<<blocks, th, shared_mem_size>>>(*in, *out, n);
    hipError_t err = hipGetLastError();
    if(err != hipSuccess){
      puts(hipGetErrorString(err));
    }
    if(*out==d1_out){
      out=&d2_out; in=&d1_out;
    }
    else{
      out=&d1_out; in=&d2_out;
    }
    n=blocks;
    // blocks=(blocks-1)/th+1;  //reduce0
    blocks=(blocks-1)/(2*th)+1;  //reduce3
    hipDeviceSynchronize();
  }
  //reduce0<<<blocks,th,shared_mem_size>>>(*in,*out,n);
  reduce3<<<blocks,th,shared_mem_size>>>(*in,*out,n);
  hipMemcpy(&result,*out,sizeof(float),hipMemcpyDeviceToHost);

  hipFree(d1_out);
  hipFree(d2_out);
  d1_out=NULL;
  d2_out=NULL;

  return result*DX*DY;
}

float volume_all(float*d_u){
  float result;

  int n=IMAX*JMAX;
  int th=WARP;
  //int blocks=(n-1)/th+1;  //reduce0
  int blocks=(n-1)/(2*th)+1;  //reduce3

  int shared_mem_size=2*th*sizeof(float);

  float *d1_out, *d2_out;
  hipMalloc((void**)&d1_out,sizeof(float)*blocks);
  hipMalloc((void**)&d2_out,sizeof(float)*blocks);

  float **in=&d_u,**out=&d1_out;
  while (blocks>1) {
    //reduce0<<<blocks, th, shared_mem_size>>>(*in, *out, n);
    reduce3<<<blocks, th, shared_mem_size>>>(*in, *out, n);
    hipError_t err = hipGetLastError();
    if(err != hipSuccess){
      puts(hipGetErrorString(err));
    }
    if(*out==d1_out){
      out=&d2_out; in=&d1_out;
    }
    else{
      out=&d1_out; in=&d2_out;
    }
    n=blocks;
    // blocks=(blocks-1)/th+1;  //reduce0
    blocks=(blocks-1)/(2*th)+1;  //reduce3
    hipDeviceSynchronize();
  }
  //reduce0<<<blocks,th,shared_mem_size>>>(*in,*out,n);
  reduce3<<<blocks, th, shared_mem_size>>>(*in,*out,n);
  hipMemcpy(&result,*out,sizeof(float),hipMemcpyDeviceToHost);

  hipFree(d1_out);
  hipFree(d2_out);
  d1_out=NULL;
  d2_out=NULL;

  return result*DX*DY;
}

int volume_i_all(int*d_u){
  int result;

  int n=IMAX*JMAX;
  int th=WARP;
  //int blocks=(n-1)/th+1;  //reduce0
  int blocks=(n-1)/(2*th)+1;  //reduce3

  int shared_mem_size=2*th*sizeof(int);

  int *d1_out, *d2_out;
  hipMalloc((void**)&d1_out,sizeof(int)*blocks);
  hipMalloc((void**)&d2_out,sizeof(int)*blocks);

  int **in=&d_u,**out=&d1_out;
  while (blocks>1) {
    reduce3_i<<<blocks, th, shared_mem_size>>>(*in, *out, n);
    hipError_t err = hipGetLastError();
    if(err != hipSuccess){
      puts(hipGetErrorString(err));
    }
    if(*out==d1_out){
      out=&d2_out; in=&d1_out;
    }
    else{
      out=&d1_out; in=&d2_out;
    }
    n=blocks;
    //blocks=(blocks-1)/th+1;  //reduce0
    blocks=(blocks-1)/(2*th)+1;  //reduce3
    hipDeviceSynchronize();
  }
  reduce3_i<<<blocks,th,shared_mem_size>>>(*in,*out,n);
  hipMemcpy(&result,*out,sizeof(int),hipMemcpyDeviceToHost);

  hipFree(d1_out);
  hipFree(d2_out);
  d1_out=NULL;
  d2_out=NULL;

  return result;
}


float volume_dev(float*d_u,int cnum){
  float result;

  int n=CIMAX*CJMAX*cnum;
  int th=WARP;
  int blocks=(n-1)/th+1;

  int shared_mem_size=2*th*sizeof(float);

  float *d1_out, *d2_out;
  hipMalloc((void**)&d1_out,sizeof(float)*blocks);
  hipMalloc((void**)&d2_out,sizeof(float)*blocks);

  float **in=&d_u,**out=&d1_out;
  while (blocks>1) {
    reduce0<<<blocks, th, shared_mem_size>>>(*in, *out, n);
    hipError_t err = hipGetLastError();
    if(err != hipSuccess){
      puts(hipGetErrorString(err));
    }
    if(*out==d1_out){
      out=&d2_out; in=&d1_out;
    }
    else{
      out=&d1_out; in=&d2_out;
    }
    n=blocks;
    blocks=(blocks-1)/th+1;
    hipDeviceSynchronize();
  }
  reduce0<<<blocks,th,shared_mem_size>>>(*in,*out,n);
  hipMemcpy(&result,*out,sizeof(float),hipMemcpyDeviceToHost);

  hipFree(d1_out);
  hipFree(d2_out);
  d1_out=NULL;
  d2_out=NULL;

  return result*DX*DY;
}


// void CoM(float**d_u,float**d_u_temp,int* d){

//   dim3 cgrids;
//   dim3 cblocks;
//   cgrids.x = CIMAX*CJMAX/WARP;
//   cgrids.y = 1;
//   cgrids.z = 1;
//   cblocks.x = WARP;
//   cblocks.y = 1;
//   cblocks.z = 1;

//   for(int n=0;n<NMAX;n++){ h_ijk[n].di=0; h_ijk[n].dj=0; }

//   float *d_ux,*d_uy;
//   hipMalloc((void**)&d_ux,sizeof(float)*CIMAX*CJMAX);
//   hipMalloc((void**)&d_uy,sizeof(float)*CIMAX*CJMAX);
//   for(int n=0;n<1;n++){
//     sumxy_Kernel<<<cgrids,cblocks>>>(d_u,d_ux,d_uy,n);
//     hipDeviceSynchronize();
//     float di=volume(d_ux)/h_cells[n].v-CIMAX*0.5; //i
//     float dj=volume(d_uy)/h_cells[n].v-CJMAX*0.5; //j
//     h_ijk[n].di=(int)di; h_ijk[n].dj=(int)dj;
//     //d[n*2+0]=ii; d[n*2+1]=jj;
//   }
//   hipMemcpyToSymbol(HIP_SYMBOL(d_ijk),&h_ijk,sizeof(PARAM::com)*NMAX);

//   CoM_Kernel<<<cgrids,cblocks>>>(d_u,d_u_temp);
//   hipDeviceSynchronize();
//   update_Kernel<<<cgrids,cblocks>>>(d_u,d_u_temp);
//   hipDeviceSynchronize();

//   hipFree(d_ux); hipFree(d_uy);
//   d_ux=NULL; d_uy=NULL;
// }

// void CoM(float**d_u,float**d_u_temp,float**d_p,float**d_p_temp,int* d,int cnum){

//   dim3 cgrids;
//   dim3 cblocks;
//   cgrids.x = CIMAX*CJMAX/WARP;
//   cgrids.y = 1;
//   cgrids.z = 1;
//   cblocks.x = WARP;
//   cblocks.y = 1;
//   cblocks.z = 1;

//   for(int n=0;n<NMAX;n++){ h_ijk[n].di=0; h_ijk[n].dj=0; }

//   float *d_ux,*d_uy;
//   hipMalloc((void**)&d_ux,sizeof(float)*CIMAX*CJMAX);
//   hipMalloc((void**)&d_uy,sizeof(float)*CIMAX*CJMAX);
//   for(int n=0;n<cnum;n++){
//     sumxy_Kernel<<<cgrids,cblocks>>>(d_u,d_ux,d_uy,n);
//     hipDeviceSynchronize();
//     float di=volume(d_ux)/h_cells[n].v-CIMAX*0.5; //i
//     float dj=volume(d_uy)/h_cells[n].v-CJMAX*0.5; //j
//     h_ijk[n].di=(int)di; h_ijk[n].dj=(int)dj;
//     //d[n*2+0]=ii; d[n*2+1]=jj;
//   }
//   hipMemcpyToSymbol(HIP_SYMBOL(d_ijk),&h_ijk,sizeof(PARAM::com)*NMAX);

//   //CoM_Kernel<<<cgrids,cblocks>>>(d_u,d_u_temp);
//   CoM_up_Kernel<<<cgrids,cblocks>>>(d_u,d_u_temp,d_p,d_p_temp);
//   hipDeviceSynchronize();
//   //update_Kernel<<<cgrids,cblocks>>>(d_u,d_u_temp,CIMAX);
//   update_up_Kernel<<<cgrids,cblocks>>>(d_u,d_u_temp,d_p,d_p_temp);
//   hipDeviceSynchronize();

//   hipFree(d_ux); hipFree(d_uy);
//   d_ux=NULL; d_uy=NULL;
// }


void input_init_cells(float *r0,int cnum){
  string str=to_string(cnum);
  string fname="src/init_cells_"+str; 
  ifstream fin(fname); 
  if(!fin.is_open()){cerr<<"ERROR:Could not open input file: init_cells"<<endl;exit(8);}
  int a=0;
  fin>>a;
  cout<<cnum<<endl;
  //#pragma omp parallel for
  for(int n=0;n<cnum;n++){
    h_cells[n].Gx=0.0f; h_cells[n].Gy=0.0f;
    fin>>r0[n]>>h_cells[n].Gx>>h_cells[n].Gy;
    //cout<<r0[n]<<" "<<h_cells[n].Gx<<" "<<h_cells[n].Gy<<endl;
    h_cells[n].cimin=h_cells[n].Gx/DX-CIMAX*0.50f;
    h_cells[n].cjmin=h_cells[n].Gy/DY-CJMAX*0.50f;
    cout<<h_cells[n].cimin<<" "<<h_cells[n].cjmin<<endl;
  }
  fin.close();
}


void time_evolution_poles(float* m_u,float* e_eta,float (&r1)[2],float (&r2)[2],int cnum,int n){

  srand((unsigned int)time(NULL));

  dim3 grids;
  dim3 blocks;
  grids.x = CIMAX*CJMAX/WARP;
  grids.y = 1;
  grids.z = 1;
  blocks.x = WARP;
  blocks.y = 1;
  blocks.z = 1;

  float r1b[2]={}; float r2b[2]={};
  float *f1x,*f1y,*f2x,*f2y;
  hipMalloc((float**)&f1x,sizeof(float)*CIMAX*CJMAX);
  hipMalloc((float**)&f1y,sizeof(float)*CIMAX*CJMAX);
  hipMalloc((float**)&f2x,sizeof(float)*CIMAX*CJMAX);
  hipMalloc((float**)&f2y,sizeof(float)*CIMAX*CJMAX);
  float F1[2]={}; float F2[2]={};

  //noise
  float theta;
  float noise1[2]={}; float noise2[2]={};

  int CFGcount=0;
  int CFGcount2=0;
  while(CFGcount<5000 && CFGcount2<50000){
    //cout<<"CFG "<<CFGcount<<endl;
    for(int d=0;d<2;d++){r1b[d]=r1[d]; r2b[d]=r2[d]; F1[d]=0; F2[d]=0;}

    force_field_Kernel<<<grids,blocks>>>(m_u,e_eta,r1[0],r1[1],r2[0],r2[1],f1x,f1y,f2x,f2y,n);
    F1[0]=volume(f1x); F1[1]=volume(f1y);
    F2[0]=volume(f2x); F2[1]=volume(f2y);
    //cout<<F1[0]<<" "<<F1[1]<<" "<<F2[0]<<" "<<F2[1]<<endl;

    //noise
    if(cnum!=1){
      theta=rand()*M_PI*2.0f;
      noise1[0]=-cosf(theta)*0.001f; noise1[1]=-sinf(theta)*0.001f;
      noise2[0]=cosf(theta+M_PI)*0.001f; noise2[1]=sinf(theta+M_PI)*0.001f;
    }

    for(int d=0;d<2;d++){
      if(CFGcount>4000 || CFGcount2>40000){
 	r1[d]+=(F1[d]-h_para.sigma*(distance_h(r1,r2)-h_para.ls)
		*((r1[d]-r2[d])/distance_h(r1,r2)))/h_para.mu*h_sys.dts;
 	r2[d]+=(F2[d]-h_para.sigma*(distance_h(r1,r2)-h_para.ls)
		*((r2[d]-r1[d])/distance_h(r1,r2)))/h_para.mu*h_sys.dts;
      }
      else{
 	// add noise 20191127----------------------- start
 	r1[d]+=(F1[d]-h_para.sigma*(distance_h(r1,r2)-h_para.ls)
		*((r1[d]-r2[d])/distance_h(r1,r2)))/h_para.mu*h_sys.dts+noise1[d];
 	r2[d]+=(F2[d]-h_para.sigma*(distance_h(r1,r2)-h_para.ls)
		*((r2[d]-r1[d])/distance_h(r1,r2)))/h_para.mu*h_sys.dts+noise2[d];
 	// add noise 20191127----------------------- end
      }
    }
    //cout<<r1[0]<<" "<<r1[1]<<" "<<r2[0]<<" "<<r2[1]<<endl;

    if(fabsf(r1b[0]-r1[0])<0.000001 && fabsf(r1b[1]-r1[1])<0.000001 &&
       fabsf(r2b[0]-r2[0])<0.000001 && fabsf(r2b[1]-r2[1])<0.000001){CFGcount++;}
    CFGcount2++;
  }

  if(CFGcount>4999) cout<<"division OK"<<endl;
  else if(CFGcount2>49999) cout<<"division error"<<endl;

  hipFree(f1x); hipFree(f1y);
  hipFree(f2x); hipFree(f2y);
  f1x=NULL; f1y=NULL;
  f2x=NULL; f2y=NULL;
}

void division(float*u,float(&r1)[2],float(&r2)[2],int m,int n){

  dim3 grids;
  dim3 blocks;
  grids.x = CIMAX*CJMAX/WARP;
  grids.y = 1;
  grids.z = 1;
  blocks.x = WARP;
  blocks.y = 1;
  blocks.z = 1;

  float *m_u_temp;
  hipMalloc((float**)&m_u_temp,sizeof(float)*CIMAX*CJMAX);
  copy_mother_cell_Kernel<<<grids,blocks>>>(u,m_u_temp,m);
  hipDeviceSynchronize();

  cout<<"set daughter cells...";
  set_daughter_cells_Kernel<<<grids,blocks>>>(m_u_temp,u,r1[0],r1[1],r2[0],r2[1],m,n);
  hipDeviceSynchronize();
  cout<<"Done."<<endl;

  hipFree(m_u_temp);
  m_u_temp=NULL;
}

void output_all_usc(int t,string Dir,float*u,float*s,float*c){

  string fnameu,fnames,fnamec;
  fnameu=Dir+"/u_"+to_string(t)+".dat";
  fnames=Dir+"/s_"+to_string(t)+".dat";
  fnamec=Dir+"/c_"+to_string(t)+".dat";

  ofstream foutu(fnameu); ofstream fouts(fnames); ofstream foutc(fnamec); 
  if(!foutu.is_open()){cerr<<"ERROR:Could not open output file u"<<endl;exit(8);}
  if(!fouts.is_open()){cerr<<"ERROR:Could not open output file s"<<endl;exit(8);}
  if(!foutc.is_open()){cerr<<"ERROR:Could not open output file c"<<endl;exit(8);}
  for(int j=JMAX-2;j>=0;j=j-2){
    for(int i=0;i<IMAX;i=i+2){
      foutu<<u[i+IMAX*j]<<" ";fouts<<s[i+IMAX*j]<<" ";foutc<<c[i+IMAX*j]<<" ";
    }
    foutu<<endl; fouts<<endl; foutc<<endl;
  }
  foutu.close(); fouts.close(); foutc.close();
}



void e_eta(float*d_e_eta,float* hd_u_m1,float*d_u,int m1,int cnum){

  string fname;
  dim3 grids;
  dim3 blocks;
  grids.x = IMAX*JMAX/WARP;
  grids.y = 1;
  grids.z = 1;
  blocks.x = WARP;
  blocks.y = 1;
  blocks.z = 1;

  float *d_u_m1;
  hipMalloc((float**)&d_u_m1,sizeof(float)*IMAX*JMAX);
  init_all_Kernel<<<grids,blocks>>>(d_u_m1);
  hipDeviceSynchronize();
  grids.x = CIMAX*CJMAX/WARP;
  u_m1_Kernel<<<grids,blocks>>>(d_u_m1,hd_u_m1,m1);
  hipDeviceSynchronize();

  for(int m2=0;m2<cnum;m2++){
    if(m2!=m1){
      e_eta_Kernel<<<grids,blocks>>>(d_e_eta,d_u,d_u_m1,m2);
      hipDeviceSynchronize();
    }
  }
  hipFree(d_u_m1);
  d_u_m1=NULL;
}

void update_lumen_pressure(float *d_u_all,float *d_s,float volume_s){
  dim3 grids;
  dim3 blocks;
  grids.x = IMAX*JMAX/WARP;
  grids.y = 1;
  grids.z = 1;
  blocks.x = WARP;
  blocks.y = 1;
  blocks.z = 1;

  //feedback control of lumen pressure 2
  float *d_overlap;
  hipMalloc((float**)&d_overlap,sizeof(float)*IMAX*JMAX);
  init_all_Kernel<<<grids,blocks>>>(d_overlap);
  hipDeviceSynchronize();
  overlap_us_Kernel<<<grids,blocks>>>(d_u_all,d_s,d_overlap);
  hipDeviceSynchronize();
  float sum_overlap=volume_all(d_overlap);

  if(sum_overlap>0.0f) h_xi=h_para.xi*sum_overlap/volume_s;
  else h_xi=0.0f;

  hipMemcpyToSymbol(HIP_SYMBOL(d_xi),&h_xi,sizeof(float));
  hipFree(d_overlap);
  d_overlap=NULL;
}


void timestamp(const int time,string Dir){
  string fname;
  fname=Dir+"/timestamp.dat";
  ofstream fout(fname.c_str()); 
  if(!fout.is_open()){cerr<<"ERROR:Could not open output: timestamp"<<fname<<endl;exit(8);}
  fout<<time<<endl;
  fout.close();
}

float h_h(float u){
  return u*u*(3.0f-2.0f*u);
}


/////////////////////////////////////////////////////////////////////////////////
//
//  main code
//
/////////////////////////////////////////////////////////////////////////////////


int main(int argc, char *argv[]){

  string fname;

  //Mersenne twister
  //my_srand(atoi(argv[1]));
  my_srand(34);

  // input parameters--------------------------------------------------start

  string Dir="DATA/"+string(argv[1]);

  int tmax   = std::round(T/DT);
  int out_dt = std::round(TOUT/DT);
  int out_dt2= std::round(TOUT2/DT);
  cout<<"tmax="<<tmax<<" out_dt="<<out_dt<<" out_dt2="<<out_dt2<<endl;

  float T_th = -1.0f;

  h_sys.dx=DX; h_sys.dy=DY;
  h_sys.imax=IMAX; h_sys.jmax=JMAX;
  h_sys.cimax=CIMAX; h_sys.cjmax=CJMAX;
  h_sys.dt =DT;
  h_sys.nmax=NMAX;
  h_sys.dts=DTS;
  hipMemcpyToSymbol(HIP_SYMBOL(d_sys),&h_sys,sizeof(PARAM::psys));

  h_para.D_u    =0.001f;
  h_para.tau_u  =1.0f;
  h_para.V      =3.0f;
  h_para.vd     =0.1f;
  h_para.alpha  =1.0f;
  h_para.beta   =1.0f;
  h_para.eta    =0.0075f;
  h_para.gamma  =0.0f;
  h_para.gamma_curv=0.015f;

  float tau_V   =atof(argv[2]);
  float noise_tau_V  =0.25f;
  h_para.alpha_V=1.00f;

  h_para.D_s    =0.001f;
  h_para.beta_s =1.0f;
  h_para.eta_s  =0.000f;
  h_para.gamma_s=0.00f;
  h_para.tau_s  =1.0f;
  h_para.xi     =atof(argv[3]);
  float xi2     =0.005f;
  h_para.alpha_s=1.0f;
  h_para.p_st   =0.8f;
  h_para.v_t    =0.35f;

  h_para.D_c    =0.001f;
  h_para.beta_cu=1.0f;
  h_para.beta_cs=1.0f;
  h_para.eta_cu =0.001f;
  h_para.gamma_c=0.00f;
  h_para.tau_c  =1.0f;
  h_para.xi_c   =0.005f;
  h_para.alpha_c=0.001f;

  h_para.ep_d   =0.100f;
  h_para.rho0   =0.010f;
  h_para.rhoe   =5.000f;
  h_para.mu     =1.000f;
  h_para.sigma  =0.001f;
  h_para.ls     =0.000f;

  h_para.D_p    =0.001f;
  h_para.tau_p  =1.0f;
  h_para.alpha_p=1.0f;
  h_para.eta_ps =0.000f;
  h_para.gamma_p=0.00f;
  h_para.Vp     =0.01f;
  h_para.C_p    =0.7f;
  h_para.w_p    =1.0f;
  h_para.p_r    =0.50f;
  h_para.k_p    =1.0f;
  h_para.p_th   =0.8f;
  float p_th2   =1.0f;
  h_para.l_anti =0.02f;

  hipMemcpyToSymbol(HIP_SYMBOL(d_para),&h_para,sizeof(PARAM::param));

  float tauV[NMAX]={};
  for(int n=0;n<NMAX;n++){
    h_cells[n].Gx=0.0f; h_cells[n].Gy=0.0f;
    h_cells[n].cimin=0; h_cells[n].cjmin=0;
    h_cells[n].v=0.0f; h_cells[n].targetv=0.0f;
    h_cells[n].vp=0.0f;
    tauV[n]=0.0f;
  }
  float volume_s=0.0f;
  float volume_c=0.0f;

  //int cnum=atoi(argv[2]);
  int cnum=8;
  hipMemcpyToSymbol(HIP_SYMBOL(d_cnum),&cnum,sizeof(int));

  float *h_r0;
  h_r0=(float*)malloc(sizeof(float)*cnum);
  for(int i=0;i<cnum;i++) h_r0[i]=0.0f;
  input_init_cells(h_r0,cnum);

  for(int n=0;n<NMAX;n++){
    h_ijk[n].di=0; h_ijk[n].dj=0;
  }

  float init_theta=0.0;

  //reshaping
  float dtau=0.04f;//<=dx*dx/sqrt(D)/4 (Olsson 2005)
  float tol=0.001f;
  // input parameters--------------------------------------------------end

  // definision--------------------------------------------------start
  dim3 cgrids(CIMAX*CJMAX/WARP,1,1), cblocks(WARP,1,1);
  dim3 ccgrids(CIMAX*CJMAX*NMAX/WARP,1,1), ccblocks(WARP,1,1);
  dim3 grids(IMAX*JMAX/WARP,1,1), blocks(WARP,1,1);

  //cells
  float *h_u,*d_u,*d_u_temp;
  h_u = (float*)malloc(sizeof(float)*NMAX*CIMAX*CJMAX);
  hipMalloc((void **)&d_u,sizeof(float)*NMAX*CIMAX*CJMAX);
  hipMalloc((void **)&d_u_temp,sizeof(float)*NMAX*CIMAX*CJMAX);
  for(int i=0;i<NMAX*CIMAX*CJMAX;i++) h_u[i]=0.0f;
  hipMemcpy(d_u,h_u,sizeof(float)*NMAX*CIMAX*CJMAX,hipMemcpyHostToDevice);
  hipMemcpy(d_u_temp,h_u,sizeof(float)*NMAX*CIMAX*CJMAX,hipMemcpyHostToDevice);

  //anti-adhesive molecules
  float *h_p,*d_p,*d_p_temp;
  h_p = (float*)malloc(sizeof(float)*NMAX*CIMAX*CJMAX);
  hipMalloc((void **)&d_p,sizeof(float)*NMAX*CIMAX*CJMAX);
  hipMalloc((void **)&d_p_temp,sizeof(float)*NMAX*CIMAX*CJMAX);
  for(int i=0;i<NMAX*CIMAX*CJMAX;i++) h_p[i]=0.0f; 
  hipMemcpy(d_p,h_p,sizeof(float)*NMAX*CIMAX*CJMAX,hipMemcpyHostToDevice);
  hipMemcpy(d_p_temp,h_p,sizeof(float)*NMAX*CIMAX*CJMAX,hipMemcpyHostToDevice);

  //lumen
  float *h_s,*d_s,*d_s_temp;
  h_s = (float *)malloc(sizeof(float)*IMAX*JMAX);
  hipMalloc((void **)&d_s,sizeof(float)*IMAX*JMAX);
  hipMalloc((void **)&d_s_temp,sizeof(float)*IMAX*JMAX);
  for(int i=0;i<IMAX*JMAX;i++) h_s[i] = 0; 
  hipMemcpy(d_s,h_s,sizeof(float)*IMAX*JMAX,hipMemcpyHostToDevice);
  hipMemcpy(d_s_temp,h_s,sizeof(float)*IMAX*JMAX,hipMemcpyHostToDevice);

  //medium
  float *h_c,*d_c,*d_c_temp;
  h_c = (float *)malloc(sizeof(float)*IMAX*JMAX);
  hipMalloc((void **)&d_c,sizeof(float)*IMAX*JMAX);
  hipMalloc((void **)&d_c_temp,sizeof(float)*IMAX*JMAX);
  for(int i=0;i<IMAX*JMAX;i++) h_c[i] = 0; 
  hipMemcpy(d_c,h_c,sizeof(float)*IMAX*JMAX,hipMemcpyHostToDevice);
  hipMemcpy(d_c_temp,h_c,sizeof(float)*IMAX*JMAX,hipMemcpyHostToDevice);

  //reshaping
  float *d_dev_u,*d_vx_u,*d_vy_u;
  hipMalloc((void **)&d_dev_u,sizeof(float)*NMAX*CIMAX*CJMAX);
  hipMalloc((void **)&d_vx_u,sizeof(float)*NMAX*CIMAX*CJMAX);
  hipMalloc((void **)&d_vy_u,sizeof(float)*NMAX*CIMAX*CJMAX);

  float *d_dev_s,*d_vx_s,*d_vy_s;
  hipMalloc((void **)&d_dev_s,sizeof(float)*IMAX*JMAX);
  hipMalloc((void **)&d_vx_s,sizeof(float)*IMAX*JMAX);
  hipMalloc((void **)&d_vy_s,sizeof(float)*IMAX*JMAX);

  float *d_dev_c,*d_vx_c,*d_vy_c;
  hipMalloc((void **)&d_dev_c,sizeof(float)*IMAX*JMAX);
  hipMalloc((void **)&d_vx_c,sizeof(float)*IMAX*JMAX);
  hipMalloc((void **)&d_vy_c,sizeof(float)*IMAX*JMAX);

  //\sum_m u_m & \sum_m p_m
  float *h_uall,*d_uall;
  h_uall = (float *)malloc(sizeof(float)*IMAX*JMAX);
  hipMalloc((float**)&d_uall,sizeof(float)*IMAX*JMAX);
  float *h_pall,*d_pall;
  h_pall = (float *)malloc(sizeof(float)*IMAX*JMAX);
  hipMalloc((float**)&d_pall,sizeof(float)*IMAX*JMAX);
  for(int i=0;i<IMAX*JMAX;i++){ h_uall[i] = 0; h_pall[i] = 0; }
  hipMemcpy(d_uall,h_uall,sizeof(float)*IMAX*JMAX,hipMemcpyHostToDevice);
  hipMemcpy(d_pall,h_pall,sizeof(float)*IMAX*JMAX,hipMemcpyHostToDevice);

  //each u & p
  float *d_u_n, *d_p_n;
  hipMalloc((float**)&d_u_n,sizeof(float)*CIMAX*CJMAX);
  hipMalloc((float**)&d_p_n,sizeof(float)*CIMAX*CJMAX);

  //\sum_m h(u_m)
  float *d_phi;
  hipMalloc((void **)&d_phi,sizeof(float)*IMAX*JMAX);

  //CFG
  float *d_e_eta;
  hipMalloc((float**)&d_e_eta,sizeof(float)*IMAX*JMAX);

  //anti-adhesion
  float *d_u_adhe;
  hipMalloc((float**)&d_u_adhe,sizeof(float)*IMAX*JMAX);

  //CoM
  float *d_ux,*d_uy;
  hipMalloc((void**)&d_ux,sizeof(float)*CIMAX*CJMAX);
  hipMalloc((void**)&d_uy,sizeof(float)*CIMAX*CJMAX);
  // definision--------------------------------------------------end


  // initial condition--------------------------------------------------start
  ccgrids.x = CIMAX*CJMAX*cnum/WARP;
  float *d_r0;
  hipMalloc((void **)&d_r0,sizeof(float)*cnum);
  hipMemcpy(d_r0,h_r0,sizeof(float)*cnum,hipMemcpyHostToDevice);

  init_u_Kernel<<<ccgrids,ccblocks>>>(d_u,d_r0);
  hipDeviceSynchronize();
  hipFree(d_r0); d_r0=NULL;


  //cell & p volume
  cgrids.y = 1;
  for(int n=0;n<cnum;n++){
    u_n_Kernel<<<cgrids,cblocks>>>(d_u_n,d_u,n);
    hipDeviceSynchronize();
    h_cells[n].v=volume_h(d_u_n);

    if(h_para.alpha_V>0.0f){
      h_dtv[n].dtargetv=h_cells[n].v*1.10;
      h_cells[n].targetv=(float)h_dtv[n].dtargetv;
    }
    else h_cells[n].targetv=h_para.V;
  }
  hipMemcpyToSymbol(HIP_SYMBOL(d_cells),&h_cells,sizeof(PARAM::cells)*NMAX);

  //target cell volume
  for(int n=0;n<cnum;n++) tauV[n]=tau_V+(MT_rand()*2.0f-1.0f)*noise_tau_V*tau_V;

  // uall&pall&phi
  init_all_uphi_Kernel<<<grids,blocks>>>(d_uall,d_phi);
  hipDeviceSynchronize();
  cgrids.y = 1;
  for(int n=0;n<cnum;n++){
    all_uphi_Kernel<<<cgrids,cblocks>>>(d_uall,d_phi,d_u,n);
    hipDeviceSynchronize();
  }

  init_all_Kernel<<<grids,blocks>>>(d_u_adhe);	  
  hipDeviceSynchronize();
  for(int m=0;m<cnum;m++){
    float *d_u_m;
    hipMalloc((float**)&d_u_m,sizeof(float)*CIMAX*CJMAX);
    u_n_Kernel<<<cgrids,cblocks>>>(d_u_m,d_u,m);
    hipDeviceSynchronize();
    e_eta(d_u_adhe,d_u_m,d_u,m,cnum);
    hipFree(d_u_m);
    d_u_m=NULL;
  }


  // medium
  init_medium_Kernel<<<grids,blocks>>>(d_c,d_uall);
  hipDeviceSynchronize();
  grids.x = IMAX/WARP;
  boundary_dirichlet_Kernel<<<grids,blocks>>>(d_c,d_c_temp);
  hipDeviceSynchronize();
  volume_c=volume_all(d_c);
  grids.x = IMAX*JMAX/WARP;

  //calculation area for lumen & medium
  int imin_sc,jmin_sc,imax_sc,jmax_sc,imin_temp,jmin_temp;
  dim3 grids_sc, blocks_sc;
  imin_sc=h_cells[0].cimin; jmin_sc=h_cells[0].cjmin;
  imin_temp=h_cells[0].cimin; jmin_temp=h_cells[0].cjmin;
  for(int n=1;n<cnum;n++){
    if(imin_sc>h_cells[n].cimin) imin_sc=h_cells[n].cimin;
    if(jmin_sc>h_cells[n].cjmin) jmin_sc=h_cells[n].cjmin;
    if(imin_temp<h_cells[n].cimin) imin_temp=h_cells[n].cimin;
    if(jmin_temp<h_cells[n].cjmin) jmin_temp=h_cells[n].cjmin;
    }
  imax_sc=(imin_temp-imin_sc)+CIMAX;
  if(imax_sc>IMAX) imax_sc=IMAX;
  jmax_sc=(jmin_temp-jmin_sc)+CJMAX;
  if(jmax_sc>JMAX) jmax_sc=JMAX;

  cout<<imin_sc<<" "<<jmin_sc<<" "<<imax_sc<<" "<<jmax_sc<<endl;
  grids_sc.x=(imax_sc*jmax_sc-1)/WARP+1;
  blocks_sc.x=WARP;
  // initial condition--------------------------------------------------end


  // output initial condition--------------------------------------------------start
  hipMemcpy(h_uall, d_uall, sizeof(float)*IMAX*JMAX, hipMemcpyDeviceToHost);
  hipMemcpy(h_c, d_c, sizeof(float)*IMAX*JMAX, hipMemcpyDeviceToHost);
  output_all_usc(0,Dir,h_uall,h_s,h_c);

  // cell volume
  fname=Dir+"/Volume.dat";
  ofstream fout_vol(fname.c_str()); 
  if(!fout_vol.is_open()){cerr<<"ERROR:Could not open output: Volume"<<endl;exit(8);}
  fout_vol<<"0 "<<volume_s<<" "<<volume_c<<" ";
  for(int n=0;n<NMAX;n++) fout_vol<<h_cells[n].v<<" "
				  <<h_dtv[n].dtargetv<<" "
				  <<h_cells[n].vp<<" ";
  fout_vol<<endl;

  //output file number of cell & center
  fname=Dir+"/ncell_t.dat";
  ofstream fout_nc(fname.c_str());
  if(!fout_nc.is_open()){cerr<<"ERROR:Could not open output: number of cell data"<<endl;exit(8);}
  fout_nc<<"0 "<<cnum<<" ";
  for(int n=0;n<NMAX;n++) fout_nc<<fixed<<setprecision(6)<<h_cells[n].Gx<<" "<<h_cells[n].Gy<<" ";
  fout_nc<<endl;
  // output initial condition--------------------------------------------------end


  // time evolution--------------------------------------------------start
  cout<<"calculation start..."<<endl;

  int tstop=0;
  for(int t=1;t+tstop<tmax+1;t++){

    if(cnum==NMAX&&h_para.alpha_s>0.1f){
      float ave_v=0.0f;
      for(int n=0;n<NMAX;n++) ave_v+=h_cells[n].v;
      ave_v=ave_v/NMAX;
      if(ave_v>2.96f){
        h_para.alpha_s=0.0f;
        hipMemcpyToSymbol(HIP_SYMBOL(d_para),&h_para,sizeof(PARAM::param));
	cout<<"alpha_s=0.0"<<endl;
      }
    }
    if(t==(int)(T_th/DT)){
      h_para.xi=xi2;
      h_para.p_th=p_th2*h_para.p_th;
      hipMemcpyToSymbol(HIP_SYMBOL(d_para),&h_para,sizeof(PARAM::param));
      cout<<"pressure decrease t="<<T_th<<endl;
    }

    // time evolution--------------------------------------------------start
    cgrids.y = cnum;
    ccgrids.x = CIMAX*CJMAX*cnum/WARP;
    time_evolution_u_with_reshaping_Kernel<<<ccgrids,ccblocks>>>(d_u_temp,d_u,
     								  d_phi,d_s,d_c);

    time_evolution_sc_with_reshaping_2_Kernel<<<grids_sc,blocks_sc>>>(d_c_temp,d_c,d_s_temp,d_s,
								      d_phi,d_uall,
								      imin_sc,jmin_sc,imax_sc,jmax_sc,
								      volume_c);
    hipDeviceSynchronize();
    // time evolution--------------------------------------------------end

    // update--------------------------------------------------start
    update_Kernel<<<ccgrids,ccblocks>>>(d_u,d_u_temp);
    update_sc_2_Kernel<<<grids_sc,blocks_sc>>>(d_s,d_s_temp,d_c,d_c_temp,
					       imin_sc,jmin_sc,imax_sc,jmax_sc);
    hipDeviceSynchronize();

    //volume of each cell & p
    cgrids.y = 1;
    for(int n=0;n<cnum;n++){
      up_n_Kernel<<<cgrids,cblocks>>>(d_u_n,d_u,d_p_n,d_p,n);
      hipDeviceSynchronize();
      h_cells[n].v=volume_h(d_u_n);

      h_dtv[n].dtargetv
	+=(double)(h_sys.dt/tauV[n]*h_para.alpha_V)*((double)h_para.V-h_dtv[n].dtargetv);

      h_cells[n].targetv=(float)h_dtv[n].dtargetv;

    }
    hipMemcpyToSymbol(HIP_SYMBOL(d_cells),&h_cells,sizeof(PARAM::cells)*NMAX);


    //CoM
    if(t%out_dt==0){
      cgrids.y = 1;
      for(int n=0;n<cnum;n++){
	sumxy_Kernel<<<cgrids,cblocks>>>(d_u,d_ux,d_uy,n);
	hipDeviceSynchronize();
	float di=volume(d_ux)/h_cells[n].v-CIMAX*0.5f; //i
	float dj=volume(d_uy)/h_cells[n].v-CJMAX*0.5f; //j
	h_ijk[n].di=(int)di; h_ijk[n].dj=(int)dj;
	h_cells[n].cimin+=(int)di; h_cells[n].cjmin+=(int)dj;
	h_cells[n].Gx+=(float)(((int)di)*DX);
	h_cells[n].Gy+=(float)(((int)dj)*DY);
      }
      hipMemcpyToSymbol(HIP_SYMBOL(d_ijk),&h_ijk,sizeof(PARAM::com)*NMAX);
      hipMemcpyToSymbol(HIP_SYMBOL(d_cells),&h_cells,sizeof(PARAM::cells)*NMAX);

      cgrids.y = cnum;
      CoM_up_Kernel<<<cgrids,cblocks>>>(d_u,d_u_temp,d_p,d_p_temp);
      hipDeviceSynchronize();
      update_Kernel<<<ccgrids,ccblocks>>>(d_u,d_u_temp);
      hipDeviceSynchronize();


      //calculation area for lumen & medium
      imin_sc=h_cells[0].cimin; jmin_sc=h_cells[0].cjmin;
      imin_temp=h_cells[0].cimin; jmin_temp=h_cells[0].cjmin;
      for(int n=1;n<cnum;n++){
	if(imin_sc>h_cells[n].cimin) imin_sc=h_cells[n].cimin;
	if(jmin_sc>h_cells[n].cjmin) jmin_sc=h_cells[n].cjmin;
	if(imin_temp<h_cells[n].cimin) imin_temp=h_cells[n].cimin;
	if(jmin_temp<h_cells[n].cjmin) jmin_temp=h_cells[n].cjmin;
      }
      imax_sc=(imin_temp-imin_sc)+CIMAX;
      if(imax_sc>IMAX) imax_sc=IMAX;
      jmax_sc=(jmin_temp-jmin_sc)+CJMAX;
      if(jmax_sc>JMAX) jmax_sc=JMAX;

      grids_sc.x=(imax_sc*jmax_sc-1)/WARP+1;
      blocks_sc.x=WARP;

      //cell reach to edge of system
      for(int n=0;n<cnum;n++){
	if(h_cells[n].cimin<1 || h_cells[n].cimin+CIMAX>IMAX-1 || 
	   h_cells[n].cjmin<1 || h_cells[n].cjmin+CJMAX>JMAX-1){
	   tstop=tmax;
	   cout<<"a cell reaches to edge of system."<<endl;
	   }
      }

    }
    // update--------------------------------------------------end


    // intermediate step--------------------------------------------------start
    cgrids.x = CIMAX*CJMAX/WARP;
    cgrids.y = cnum;
    cblocks.x = WARP;
    normal_vector_field_Kernel<<<ccgrids,ccblocks>>>(d_vx_u,d_vy_u,d_u);
    hipDeviceSynchronize();

    cgrids.x = CIMAX/32;
    cblocks.x = 32;
    boundary_normal_vector_field_Kernel<<<cgrids,cblocks>>>(d_vx_u,d_vy_u);
    hipDeviceSynchronize();

    cgrids.x = CIMAX*CJMAX/WARP;
    cblocks.x = WARP;
    int count=0;
    float dev=cnum*tol*dtau*2.0f;
    while(dev>cnum*tol*dtau){
      reshaping_Kernel<<<ccgrids,ccblocks>>>(d_vx_u,d_vy_u,d_u,d_u_temp,dtau);
      hipDeviceSynchronize();

      dev_Kernel<<<cgrids,cblocks>>>(d_dev_u,d_u,d_u_temp);
      hipDeviceSynchronize();

      dev=0.0f;
      for(int n=0;n<cnum;n++){
     	u_n_Kernel<<<cgrids,cblocks>>>(d_u_n,d_dev_u,n);
     	hipDeviceSynchronize();
     	dev+=volume(d_u_n);
      }

      cgrids.y = cnum;
      update_Kernel<<<ccgrids,ccblocks>>>(d_u,d_u_temp);
      hipDeviceSynchronize();
      count++;
    }

    // reshaping s & c
    if(volume_s>0.0f){
      normal_vector_field_sc_2_Kernel<<<grids_sc,blocks_sc>>>(d_vx_s,d_vy_s,d_s,d_vx_c,d_vy_c,d_c,
							      imin_sc,jmin_sc,imax_sc,jmax_sc);
      hipDeviceSynchronize();
      grids.x = IMAX/WARP;
      if(imin_sc==0 || jmin_sc==0 || imax_sc==IMAX-1 || jmax_sc==JMAX-1)
	boundary_normal_vector_field_sc_Kernel<<<grids,blocks>>>(d_vx_s,d_vy_s,d_vx_c,d_vy_c);
      hipDeviceSynchronize();
    }
    else{
      normal_vector_field_all_2_Kernel<<<grids_sc,blocks_sc>>>(d_vx_c,d_vy_c,d_c,imin_sc,jmin_sc,imax_sc,jmax_sc);
      hipDeviceSynchronize();
      grids.x = IMAX/WARP;
      if(imin_sc==0 || jmin_sc==0 || imax_sc==IMAX-1 || jmax_sc==JMAX-1)
	boundary_normal_vector_field_all_Kernel<<<grids,blocks>>>(d_vx_c,d_vy_c);
      hipDeviceSynchronize();
    }

    grids.x = IMAX*JMAX/WARP;
    count=0;
    dev=tol*dtau*2.0f;

    if(volume_s>0.0f){
      while(dev>tol*dtau){
	reshaping_all_2_Kernel<<<grids_sc,blocks_sc>>>(d_vx_s,d_vy_s,d_s,d_s_temp,dtau,h_para.D_s,
						       imin_sc,jmin_sc,imax_sc,jmax_sc);
	hipDeviceSynchronize();

	dev_all_2_Kernel<<<grids_sc,blocks_sc>>>(d_dev_s,d_s,d_s_temp,imin_sc,jmin_sc,imax_sc,jmax_sc);
	hipDeviceSynchronize();

	dev=volume_all(d_dev_s);

	update_all_2_Kernel<<<grids_sc,blocks_sc>>>(d_s,d_s_temp,imin_sc,jmin_sc,imax_sc,jmax_sc);
	hipDeviceSynchronize();

	count++;
      }
    }

    count=0;
    dev=tol*dtau*2.0f;
    while(dev>tol*dtau){
      reshaping_all_2_Kernel<<<grids_sc,blocks_sc>>>(d_vx_c,d_vy_c,d_c,d_c_temp,dtau,h_para.D_c,
						     imin_sc,jmin_sc,imax_sc,jmax_sc);
      hipDeviceSynchronize();

      dev_all_2_Kernel<<<grids_sc,blocks_sc>>>(d_dev_c,d_c,d_c_temp,imin_sc,jmin_sc,imax_sc,jmax_sc);
      hipDeviceSynchronize();

      dev=volume_all(d_dev_c);

      update_all_2_Kernel<<<grids_sc,blocks_sc>>>(d_c,d_c_temp,imin_sc,jmin_sc,imax_sc,jmax_sc);
      hipDeviceSynchronize();

      count++;
    }
    // intermediate step--------------------------------------------------end


    // cell division---------------------------------------------------start
    bool e_eta_ON=false;
    for(int n=0;n<cnum;n++){

      if(h_cells[n].v>h_para.V-h_para.vd && cnum<NMAX){
	cgrids.y = 1;
	u_n_Kernel<<<cgrids,cblocks>>>(d_u_n,d_u,n);
	hipDeviceSynchronize();

 	if(e_eta_ON==false){
	  init_all_Kernel<<<grids,blocks>>>(d_e_eta);	  
	  for(int m=0;m<cnum;m++){
	    float *d_u_m;
	    hipMalloc((float**)&d_u_m,sizeof(float)*CIMAX*CJMAX);
	    u_n_Kernel<<<cgrids,cblocks>>>(d_u_m,d_u,m);
	    hipDeviceSynchronize();
	    e_eta(d_e_eta,d_u_m,d_u,m,cnum);
	    hipFree(d_u_m);
	    d_u_m=NULL;
	  }

 	  e_eta_ON=true;
 	}

 	//cell division
	h_cells[cnum]=h_cells[n];

	//set the angle of the division plane
	float theta=0.0f;
	if(cnum==1 && init_theta>=0.0f)
	  theta=(init_theta+90.0f)/360.0f*M_PI*2.0f;
	else
	  theta=MT_rand()*M_PI*2.0f;

	float r1[2]={}; float r2[2]={};
	r1[0]=h_cells[n].Gx-cos(theta)*h_sys.dx; r1[1]=h_cells[n].Gy-sin(theta)*h_sys.dy;
	r2[0]=h_cells[n].Gx+cos(theta)*h_sys.dx; r2[1]=h_cells[n].Gy+sin(theta)*h_sys.dy;

	time_evolution_poles(d_u_n,d_e_eta,r1,r2,cnum,n);

	float Pc[2]={};
	Pc[0]=(float)(r1[0]+r2[0])*0.50f; Pc[1]=(float)(r1[1]+r2[1])*0.50f;
	set_seed_s_Kernel<<<cgrids,cblocks>>>(d_u,d_s,Pc[0],Pc[1],n);
	
	grids.y = cnum;
	set_daughter_cells_Kernel<<<grids,blocks>>>(d_u,d_u_temp,r1[0],r1[1],r2[0],r2[1],n,cnum);
	hipDeviceSynchronize();
	hipMemcpy(d_u,d_u_temp,sizeof(float)*NMAX*CIMAX*CJMAX,hipMemcpyDeviceToDevice);

	//volume
	u_n_Kernel<<<cgrids,cblocks>>>(d_u_n,d_u,n);
	hipDeviceSynchronize();
	h_cells[n].v=volume_h(d_u_n);

	u_n_Kernel<<<cgrids,cblocks>>>(d_u_n,d_u,cnum);
	hipDeviceSynchronize();
	h_cells[cnum].v=volume_h(d_u_n);

	cout<<h_cells[n].v<<" "<<h_cells[cnum].v<<endl;
	cout<<h_cells[n].vp<<" "<<h_cells[cnum].vp<<endl;

	if(h_para.alpha_V>0.0f){
          h_dtv[cnum].dtargetv=h_cells[cnum].v*1.10f;
          h_dtv[n].dtargetv=h_cells[n].v*1.10f;
	  h_cells[cnum].targetv=(float)h_dtv[cnum].dtargetv;
	  h_cells[n].targetv=(float)h_dtv[n].dtargetv;
	}
	else{h_cells[n].targetv=h_para.V; h_cells[cnum].targetv=h_para.V;}

	hipMemcpyToSymbol(HIP_SYMBOL(d_cells),&h_cells,sizeof(PARAM::cells)*NMAX);

	tauV[n]=tau_V+(MT_rand()*2.0f-1.0f)*noise_tau_V*tau_V;
	tauV[cnum]=tau_V+(MT_rand()*2.0f-1.0f)*noise_tau_V*tau_V;

	cnum++;
	hipMemcpyToSymbol(HIP_SYMBOL(d_cnum),&cnum,sizeof(int));
      }
    }
    // cell division---------------------------------------------------end

    //uall&phi
    init_all_uphi_Kernel<<<grids,blocks>>>(d_uall,d_phi);
    hipDeviceSynchronize();
    cgrids.y = 1;
    for(int n=0;n<cnum;n++){
      all_uphi_Kernel<<<cgrids,cblocks>>>(d_uall,d_phi,d_u,n);
      hipDeviceSynchronize();
    }

    init_all_Kernel<<<grids,blocks>>>(d_u_adhe);	  
    hipDeviceSynchronize();
    for(int m=0;m<cnum;m++){
      float *d_u_m;
      hipMalloc((float**)&d_u_m,sizeof(float)*CIMAX*CJMAX);
      u_n_Kernel<<<cgrids,cblocks>>>(d_u_m,d_u,m);
      hipDeviceSynchronize();
      e_eta(d_u_adhe,d_u_m,d_u,m,cnum);
      hipFree(d_u_m);
      d_u_m=NULL;
    }

    volume_s=volume_all(d_s);
    volume_c=volume_all(d_c);

    if(t%out_dt==0){
      int t_real=t*DT;
      timestamp(t_real,Dir);

      //phase field
      hipMemcpy(h_uall, d_uall, sizeof(float)*IMAX*JMAX, hipMemcpyDeviceToHost);
      hipMemcpy(h_s, d_s, sizeof(float)*IMAX*JMAX, hipMemcpyDeviceToHost);
      hipMemcpy(h_c, d_c, sizeof(float)*IMAX*JMAX, hipMemcpyDeviceToHost);
      output_all_usc(t_real,Dir,h_uall,h_s,h_c);

      //volume
      fout_vol<<t_real<<" "<<volume_s<<" "<<volume_c<<" ";
      for(int n=0;n<NMAX;n++) fout_vol<<h_cells[n].v<<" "
				      <<h_dtv[n].dtargetv<<" "
				      <<h_cells[n].vp<<" ";
      fout_vol<<endl;

      //number of cells
      fout_nc<<t_real<<" "<<cnum<<" ";
      for(int n=0;n<NMAX;n++) fout_nc<<fixed<<setprecision(6)<<h_cells[n].Gx<<" "<<h_cells[n].Gy<<" ";
      fout_nc<<endl;
    }
  }
  // time evolution--------------------------------------------------end

  free(h_u); hipFree(d_u); hipFree(d_u_temp);
  free(h_p); hipFree(d_p); hipFree(d_p_temp);
  h_u=NULL; d_u=NULL; d_u_temp=NULL;
  h_p=NULL; d_p=NULL; d_p_temp=NULL;

  hipFree(d_u_n); hipFree(d_p_n);
  d_u_n=NULL; d_p_n=NULL;

  hipFree(d_phi);   d_phi=NULL;
  free(h_uall); hipFree(d_uall);
  h_uall=NULL; d_uall=NULL;
  free(h_pall); hipFree(d_pall);
  h_pall=NULL; d_pall=NULL;

  free(h_s); hipFree(d_s); hipFree(d_s_temp);
  h_s=NULL; d_s=NULL; d_s_temp=NULL;

  free(h_c); hipFree(d_c); hipFree(d_c_temp);
  h_c=NULL; d_c=NULL; d_c_temp=NULL;

  hipFree(d_ux); hipFree(d_uy);
  d_ux=NULL; d_uy=NULL;

  hipFree(d_dev_s); hipFree(d_vx_s); hipFree(d_vy_s);
  hipFree(d_dev_c); hipFree(d_vx_c); hipFree(d_vy_c);
  d_dev_s=NULL; d_vx_s=NULL; d_vy_s=NULL;
  d_dev_c=NULL; d_vx_c=NULL; d_vy_c=NULL;

  hipFree(d_u_adhe);
  d_u_adhe=NULL;

  fout_vol.close();
  fout_nc.close();

  return 0;
}
